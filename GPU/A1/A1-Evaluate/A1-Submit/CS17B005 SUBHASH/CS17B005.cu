
#include <hip/hip_runtime.h>


__global__ void per_row_kernel(int m,int n,int *A,int *B,int *C){
    // int id = blockIdx.x*blockDim.x+threadIdx.x;
    int td = threadIdx.z*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
    int dim = blockDim.x*blockDim.y*blockDim.z;
    int bk = blockIdx.z*gridDim.x*gridDim.y + blockIdx.y*gridDim.x + blockIdx.x;
    int id = bk*dim + td;
    if (id < m){
        for(int i = 0;i < n;++i){
            C[id*n+i] = A[id*n+i]+B[id*n+i];
        }
    }
}

__global__ void per_column_kernel(int m,int n,int *A,int *B,int *C){
    // int id = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
    int td = threadIdx.z*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
    int dim = blockDim.x*blockDim.y*blockDim.z;
    int bk = blockIdx.z*gridDim.x*gridDim.y + blockIdx.y*gridDim.x + blockIdx.x;
    int id = bk*dim + td;
    if (id < n){
        for(int i = 0;i < m;++i){
            C[i*n+id] = A[i*n+id] + B[i*n+id];
        }
    }

}

__global__ void per_element_kernel(int m,int n,int *A,int *B,int *C){
    int td = threadIdx.z*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
    int dim = blockDim.x*blockDim.y*blockDim.z;
    int bk = blockIdx.z*gridDim.x*gridDim.y + blockIdx.y*gridDim.x + blockIdx.x;
    int id = bk*dim + td;
    int x = id/n;
    if (x < m){
        int y = id%n;
        C[x*n+y] = A[x*n+y] + B[x*n+y];
    }
}
