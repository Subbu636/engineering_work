// Run: nvcc -arch=sm_50 -Wno-deprecated-gpu-targets main.cu

// Run: create object files with -c tag for both main and cs
// Now compile both obj files together: nvcc -arch=sm_50 -Wno-deprecated-gpu-targets CS17B005.obj main.obj
// run the excutable produced

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "kernels.h"

void print_matrix(int m,int n, int *mat){
    for(int i = 0;i < m;++i){
        for(int j = 0;j < n;++j){
            printf("%d ",mat[i*n+j]);
        }
        printf("\n");
    }
}

#define N 1024
int main() {
    // Sample Generated Input
    // const int m = 15,n = 10;
    // int A[n*m],B[n*m],C[n*m];
    // for(int i = 0;i < m*n;++i){
    //     A[i] = (i+1);
    //     B[i] = -1*(i);
    // }

    // Reading from files
    int m,n;
    const char input_file_name[100] = "testcases\\input\\input3.txt"; // windows file path!!
    FILE *file;
    if ((file = fopen (input_file_name, "r")) == NULL){
        printf("Cannot Open File!");
        exit(1);
    }
    fscanf (file, "%d", &m);
    fscanf (file, "%d", &n);
    int l = m*n,v;
    int *A = (int*)malloc(l*sizeof(int));
    int *B = (int*)malloc(l*sizeof(int));
    int *C = (int*)malloc(l*sizeof(int));
    for (int i = 0;i < l && !feof (file);++i){
        fscanf (file, "%d", &v);
        A[i] = v;
    }
    for (int i = 0;i < l && !feof (file);++i){
        fscanf (file, "%d", &v);
        B[i] = v;
    }
    fclose(file);

    // Core Code
    int *gpuA,*gpuB,*gpuC;
    hipMalloc(&gpuA,m*n*sizeof(int));
    hipMalloc(&gpuB,m*n*sizeof(int));
    hipMemcpy(gpuA,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpuB,B,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMalloc(&gpuC,m*n*sizeof(int));
    per_element_kernel<<<dim3(3963,1,1),dim3(328,1,1)>>>(m,n,gpuA,gpuB,gpuC);
    hipMemcpy(C,gpuC,m*n*sizeof(int),hipMemcpyDeviceToHost);
    // print_matrix(m,n,C);

    // Checking it with output
    const char output_file_name[100] = "testcases\\output\\output3.txt"; // windows file path!!
    if ((file = fopen (output_file_name, "r")) == NULL){
        printf("Cannot Open File!");
        exit(1);
    }
    bool out = true;
    for (int i = 0;i < l && !feof (file);++i){
        fscanf (file, "%d", &v);
        if(v != C[i]){
            out = false;
            break;
        }
    }
    fclose(file);
    if (out){
        printf("Correct Output");
    }
    else{
        printf("Wrong Output");
    }
    return 0;
}