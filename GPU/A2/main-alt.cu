#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include <sys/time.h> 
#include<bits/stdc++.h>
using namespace std;

struct edgepairs{
  int x;
  int y;
};

bool compareTwoEdgePairs(edgepairs a, edgepairs b)
{
    if (a.x != b.x)
        return a.x < b.x;

    if (a.y != b.y)
        return a.y < b.y;
 
  return true;
}

void print_matrix(int m,int n, int *mat){
    for(int i = 0;i < m;++i){
        for(int j = 0;j < n;++j){
            printf("%d ",mat[i*n+j]);
        }
        printf("\n");
    }
}

// complete the following kernel...
__global__ void dkernel_Adds(int *gpuOA, int *gpuCA, int *gpulocals,int *gpucurrentupdate){
	int n = gridDim.x;
	int id = blockIdx.x;
	// printf("%d,%d\n",id,n);
	if(id >= n) return;
	int var = 0;
	for(int i = gpuOA[id];i < gpuOA[id+1];++i){
		var+=gpucurrentupdate[gpuCA[i]];
	}
	__syncthreads();
	gpulocals[id] += var;
}

// complete the following kernel...
__global__ void dkernel_Mins(int *gpuOA, int *gpuCA, int *gpulocals,int *gpucurrentupdate){
	int n = gridDim.x;
	int id = blockIdx.x;
	if(id >= n) return;
	int var = gpulocals[id];
	for(int i = gpuOA[id];i < gpuOA[id+1];++i){
		var = min(var,gpucurrentupdate[gpuCA[i]]);
	}
	__syncthreads();
	gpulocals[id] = var;

}

// complete the following kernel...
__global__ void dkernel_Maxs(int *gpuOA, int *gpuCA, int *gpulocals,int *gpucurrentupdate){
	int n = gridDim.x;
	int id = blockIdx.x;
	if(id >= n) return;
	int var = gpulocals[id];
	for(int i = gpuOA[id];i < gpuOA[id+1];++i){
		var = max(var,gpucurrentupdate[gpuCA[i]]);
	}
	__syncthreads();
	gpulocals[id] = var;
}

int main(int argc,char **argv){

	//variable declarations
	int m,n;
	int number;
	int numofquery;
	int op;
	struct timeval t1, t2;
	vector <double> kerneltime;

	//File pointer declaration
	FILE *filePointer;

	//File Opening for read
	char *filename = argv[1]; 
    	filePointer = fopen( filename , "r") ; 
      
	//checking if file ptr is NULL
    	if ( filePointer == NULL ) 
    	{
        printf( "input.txt file failed to open." ) ; 
	      return 0;
    	}

	fscanf(filePointer, "%d", &n );		//scaning the number of vertices
        fscanf(filePointer, "%d", &m );		//scaning the number of edges

	//D.S to store the input graph in COO format
	vector <edgepairs> COO(m);
	
	//Reading from file and populate the COO
	for(int i=0 ; i<m ; i++ )
        {
		for(int j=0;j<2;j++){
			if ( fscanf(filePointer, "%d", &number) != 1)
            			break;
		if( j%2 == 0) 
		{       		
			if(number >= 1 && number <= 10000)
			COO[i].y = number;
		}		
		else
		{
			if(number >= 1 && number <= 10000)
			COO[i].x = number;
		}	

		}
        }
	// COO done...
	
	// sort the COO 
	sort(COO.begin(),COO.end(),compareTwoEdgePairs);
	//sorting COO done..
	
	// Converting the graph in COO format to CSR format..
	
	// create the CSR
	
	int *OA = (int *)malloc( (n+1)*sizeof(int));		//Offsets Array
	for(int i=0;i<n+1;i++){
                OA[i] = 0;
        }

	int *CA = (int *)malloc(m*sizeof(int));			//Coordinates Array
	OA[0]=0;

	//initialize the Coordinates Array
	for(int i=0;i<m;i++){
		if(COO[i].y >= 1 && COO[i].y <= 10000)
		CA[i] = COO[i].y - 1;
	}
	//initialize the Offsets Array
	for(int i=0;i<m;i++){
		if(COO[i].x >= 1 && COO[i].x <= 10000)
		OA[COO[i].x]++;		//store the frequency..
	}
	for(int i=0;i<n;i++){
		OA[i+1] += OA[i];	// do cumulative sum..
	}

	// Converting the graph to CSR done..
	
	// copy initial local values to the array from the file
	int *initlocalvals = (int *)malloc(n*sizeof(int));;
	for(int i=0 ; i<n ; i++ )
        {
        if ( fscanf(filePointer, "%d", &number) != 1)
            break;
         
        initlocalvals[i] = number;
        }
	// copying local vals end..

	// get number of queries from the file
	fscanf(filePointer, "%d", &numofquery);
	
	//copy OA,CA and initlocalvals to the GPU Memory
	int *gpuOA, *gpuCA, *gpulocals;
  hipMalloc( &gpuOA, sizeof(int) * (1+n) );
  hipMalloc( &gpuCA, sizeof(int) * m );
  hipMalloc( &gpulocals, sizeof(int) * n );
	hipMemcpy(gpuOA, OA, sizeof(int) * (1+n), hipMemcpyHostToDevice);
	hipMemcpy(gpuCA, CA, sizeof(int) * m, hipMemcpyHostToDevice);
	hipMemcpy(gpulocals, initlocalvals, sizeof(int) * n, hipMemcpyHostToDevice);

	printf("%d %d\n",n,m);
	print_matrix(1,n+1, OA);
	print_matrix(1,m,CA);
	print_matrix(1,n,initlocalvals);
	printf("\n");

	int *currentupdate = (int *)malloc(n*sizeof(int));	// array to store the updates that are pushed by each vertex to there neighbors
	int *gpucurrentupdate;		// same as above but on GPU
  hipMalloc( &gpucurrentupdate, sizeof(int) * n );
  int *results = (int *)malloc(n*sizeof(int));         // storing the results from GPU to CPU for the enumerate query


  // open the output.txt to write the query results
      // char *fname = argv[2]; 
      FILE *fptr;
      // fptr = fopen(fname,"w");
      fptr = stdout;

	for(int i=0;i<numofquery;i++){

		//read the operator
		fscanf(filePointer, "%d", &op);

		if(op != 3){					// if operator is other then enumerate (i.e. +,min,max)

		// read the current updates in the array				
			for(int j=0 ; j<n ; j++ )
	        	{
	        	 	if ( fscanf(filePointer, "%d", &number) != 1)
	            	 	break;
	        		currentupdate[j] = number;
	        	}

		// copy current updates to gpu
		hipMemcpy(gpucurrentupdate, currentupdate, sizeof(int) * n, hipMemcpyHostToDevice);
		//kernel launches
    if(op == 0)	{
		gettimeofday(&t1, 0);	
		dkernel_Adds<<<n,1>>>(gpuOA,gpuCA,gpulocals,gpucurrentupdate);
		hipDeviceSynchronize();
		gettimeofday(&t2, 0);
		}
    if(op == 1)	{
		gettimeofday(&t1, 0);
		dkernel_Mins<<<n,1>>>(gpuOA,gpuCA,gpulocals,gpucurrentupdate);
		hipDeviceSynchronize();
		gettimeofday(&t2, 0);
		}
    if(op == 2)	{	
		gettimeofday(&t1, 0);
		dkernel_Maxs<<<n,1>>>(gpuOA,gpuCA,gpulocals,gpucurrentupdate);
		hipDeviceSynchronize();
		gettimeofday(&t2, 0);
		}
    
		double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0; // Time taken by kernel in seconds 
		kerneltime.push_back(time);  

    		printf("Time taken by kernel to execute is: %.6f ms\n", time); 
		}

		else{						// if operator is enumnerate then store the results to file
			//print local values of each vertices.
      hipMemcpy(results, gpulocals, n * sizeof(int), hipMemcpyDeviceToHost);  // get each locals from GPU
       for(int j=0;j<n;j++){
           fprintf(fptr ,"%d ", results[j] );
       }
       fprintf(fptr,"\n");
    }
		
	}

	int nall = kerneltime.size();
	double sumtime=0;
	for(int i=0;i<nall;i++){
		sumtime += kerneltime[i];
	}
	// print the time taken by all the kernels of the current test-case
	cout << "\ntotal time taken by the current test-case is " << sumtime << " ms\n";

  fclose(fptr);
  fclose(filePointer);

	return 0;
}
