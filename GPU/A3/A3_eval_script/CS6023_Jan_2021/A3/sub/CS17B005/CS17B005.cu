#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>
#include<thrust/reduce.h>
using namespace std;

struct pairs{
    float first;
    int second;
};

__host__ __device__ bool compareTimes(pairs i1, pairs i2)
{
    if (i1.first == i2.first){
        return i1.second < i2.second;
    }
    return i1.first < i2.first;
}

__global__ void toll_booths(pairs *times,int n,int m,int x){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if (id < m){
        float last = times[id].first;
        for(int q = id;q < n;q+=m){
            times[q].first = max(last,times[q].first)+(float)x;
            last = times[q].first;
        }
    }
}

__global__ void travel(pairs *times,int n,int i,float dis,float *speed){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if (id < n){
        times[id].first = times[id].first+(dis/speed[((i+1)*n)+times[id].second]);
    }
}

//Complete the following function
void operations_gpu ( int n, int k, int m, int x, float dis, float *speed, int **results )  {
    pairs *times = (pairs*) malloc(n*sizeof(pairs));
    pairs *gpu_times;
    hipMalloc(&gpu_times,n*sizeof(pairs));
    float *gpu_speed;
    hipMalloc(&gpu_speed,n*(k+1)*sizeof(float));
    hipMemcpy(gpu_speed,speed,n*(k+1)*sizeof(float),hipMemcpyHostToDevice);
    for(int j = 0;j < n;++j){
        times[j].first = (dis/speed[j]);
        times[j].second = j;
    }
    hipMemcpy(gpu_times,times,n*sizeof(pairs),hipMemcpyHostToDevice);
    for(int i = 0;i < k;++i){
        thrust::sort(times,times+n,compareTimes);
        hipDeviceSynchronize();
        results[0][i] = times[0].second+1;
        results[1][i] = times[n-1].second+1;
        hipMemcpy(gpu_times,times,n*sizeof(pairs),hipMemcpyHostToDevice);
        toll_booths<<<m,1>>>(gpu_times,n,m,x);
        hipDeviceSynchronize();
        travel<<<n,1>>>(gpu_times,n,i,dis,gpu_speed);
        hipDeviceSynchronize();
        hipMemcpy(times,gpu_times,n*sizeof(pairs),hipMemcpyDeviceToHost);
    }
    thrust::sort(times,times+n,compareTimes);
    for(int j = 0;j < n;++j){
        results[2][times[j].second] = (int)times[j].first;
    }
    results[0][k] = times[0].second+1;
    results[1][k] = times[n-1].second+1;
    return;
}


// void operations_cpu ( int n, int k, int m, int x, float dis, float *speed, int **results )  {
//     pairs *times = (pairs*) malloc(n*sizeof(pairs));
//     for(int j = 0;j < n;++j){
//         times[j].first = 0.0;
//         times[j].second = j;
//     }
//     for(int i = 0;i < k;++i){
//         for(int j = 0;j < n;++j){
//             times[j].first = times[j].first+(dis/speed[(i*n)+times[j].second]);
//         }
//         sort(times,times+n,compareTimes);
//         results[0][i] = times[0].second+1;
//         results[1][i] = times[n-1].second+1;
//         for(int p = 0;p < m;++p){
//             float last = times[p].first;
//             for(int q = p;q < n;q+=m){
//                 times[q].first = max(last,times[q].first)+(float)x;
//                 last = times[q].first;
//             }
//         }
//     }
//     for(int j = 0;j < n;++j){
//         times[j].first = times[j].first+(dis/speed[(k*n)+times[j].second]);
//         results[2][times[j].second] = (int)times[j].first;
//     }
//     sort(times,times+n,compareTimes);
//     results[0][k] = times[0].second+1;
//     results[1][k] = times[n-1].second+1;
//     return;
// }

int main(int argc,char **argv){

    //variable declarations
    int n,k,m,x;
    float dis;
    
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0;
    }
    
    
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of vehicles
    fscanf( inputfilepointer, "%d", &k );      //scaning for number of toll tax zones
    fscanf( inputfilepointer, "%d", &m );      //scaning for number of toll tax points
    fscanf( inputfilepointer, "%d", &x );      //scaning for toll tax zone passing time
    
    fscanf( inputfilepointer, "%f", &dis );    //scaning for distance between two consecutive toll tax zones


    // scanning for speeds of each vehicles for every subsequent toll tax combinations
    float *speed = (float *) malloc ( n*( k+1 ) * sizeof (float) );
    for ( int i=0; i<=k; i++ )  {
        for ( int j=0; j<n; j++ )  {
            fscanf( inputfilepointer, "%f", &speed[i*n+j] );
            speed[i*n+j] = speed[i*n+j]/60.0;
        }
    }
    
    // results is in the format of first crossing vehicles list, last crossing vehicles list 
    //               and total time taken by each vehicles to pass the highway
    int **results = (int **) malloc ( 3 * sizeof (int *) );
    results[0] = (int *) malloc ( (k+1) * sizeof (int) );
    results[1] = (int *) malloc ( (k+1) * sizeof (int) );
    results[2] = (int *) malloc ( (n) * sizeof (int) );


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);


    // Function given to implement
    operations_gpu ( n, k, m, x, dis, speed, results );


    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    // First crossing vehicles list
    for ( int i=0; i<=k; i++ )  {
        fprintf( outputfilepointer, "%d ", results[0][i]);
    }
    fprintf( outputfilepointer, "\n");


    //Last crossing vehicles list
    for ( int i=0; i<=k; i++ )  {
        fprintf( outputfilepointer, "%d ", results[1][i]);
    }
    fprintf( outputfilepointer, "\n");


    //Total time taken by each vehicles to pass the highway
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", results[2][i]);
    }
    fprintf( outputfilepointer, "\n");

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    return 0;
}