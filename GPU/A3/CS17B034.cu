#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<stdio.h>
#include <thrust/sort.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>


using namespace std;


//Complete the following function
struct Car{
    int id;
    float zonetime;
    
};
struct CarSorter1
{
   __host__ __device__ bool operator()(const Car& a, const Car& b)
  {
    if (a.zonetime != b.zonetime) return a.zonetime < b.zonetime;
    else return a.id < b.id;
  }
};


struct CarSorter2
{
   __host__ __device__ bool operator()(const Car& a, const Car& b)
  {
    return a.id < b.id;
  }
};

__global__ void travelkernel(Car* gcars,int* speeds, int n, int dis)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
    {
        gcars[id].zonetime +=   ((float)dis*60)/(float)speeds[gcars[id].id];
    }
}

__global__ void waitingkernel(Car* gcars, int n,int m,int x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < m)
    {
        
        for(int j=id;j<n;j+=m)
        {
            if(j<m)
            {
                gcars[j].zonetime = gcars[j].zonetime + (float) x;
            }
            else
            {
                if(gcars[j-m].zonetime >= gcars[j].zonetime)
                {
                        gcars[j].zonetime = gcars[j-m].zonetime + (float)x ;
                }
                else gcars[j].zonetime = gcars[j].zonetime + (float)x ;
            }
        }
    }
}
__global__ void initkernel(Car* gcars,int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
    {
        gcars[id].zonetime = 0;
        gcars[id].id = id;
    }
}
__global__ void printcar(Car* gcars,int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
    {
        printf("%d, %f\n", gcars[id].id,gcars[id].zonetime);
    }
}
__global__ void printspeed(int* speeds,int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
    {
        printf("%d, %d\n",id, speeds[id]);
    }
}
void operations ( int n, int k, int m, int x, int dis, int *speed, int **results )  {
         Car cars[n];
         Car *gcars;
         hipMalloc(&gcars, n* sizeof(Car));
         int blocks = (n/1024) + 1;
        initkernel<<<blocks,1024>>>(gcars,n);
        hipDeviceSynchronize();
        for (int i=0; i<=k;i++)
        {
           // printf("%d\n",i);
            int *gspeeds;
            hipMalloc(&gspeeds, n* sizeof(int)); 
            int* offspeed = (int*) malloc(n*sizeof(int));
            for(int j=0;j<n;j++){offspeed[j] = speed[i*n+j];}
            hipMemcpy(gspeeds,offspeed,n*sizeof(int),hipMemcpyHostToDevice);

         //   printspeed<<<blocks,1024>>>(gspeeds,n);
         //   hipDeviceSynchronize();
            travelkernel<<<blocks,1024>>>(gcars,gspeeds,n,dis);
            hipMemcpy(cars,gcars,n*sizeof(Car),hipMemcpyDeviceToHost);

            sort(cars,cars+n,CarSorter1());
            
            results[0][i] = cars[0].id + 1;
            results[1][i] = cars[n-1].id + 1;
            
            if(i==k) break;
            hipMemcpy(gcars,cars,n*sizeof(Car),hipMemcpyHostToDevice);
            
            waitingkernel<<<1,m>>>(gcars,n,m,x);
            hipDeviceSynchronize();
        }
        for(int i=0;i<n;i++)
        {
            results[2][cars[i].id] = floor(cars[i].zonetime);
        }
  
}

int main(int argc,char **argv){

    //variable declarations
    int n,k,m,x;
    int dis;
    
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0;
    }
    
    
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of vehicles
    fscanf( inputfilepointer, "%d", &k );      //scaning for number of toll tax zones
    fscanf( inputfilepointer, "%d", &m );      //scaning for number of toll tax points
    fscanf( inputfilepointer, "%d", &x );      //scaning for toll tax zone passing time
    
    fscanf( inputfilepointer, "%d", &dis );    //scaning for distance between two consecutive toll tax zones


    // scanning for speeds of each vehicles for every subsequent toll tax combinations
    int *speed = (int *) malloc ( n*( k+1 ) * sizeof (int) );
    for ( int i=0; i<=k; i++ )  {
        for ( int j=0; j<n; j++ )  {
            fscanf( inputfilepointer, "%d", &speed[i*n+j] );
        }
    }
    
    // results is in the format of first crossing vehicles list, last crossing vehicles list 
    //               and total time taken by each vehicles to pass the highway
    int **results = (int **) malloc ( 3 * sizeof (int *) );
    results[0] = (int *) malloc ( (k+1) * sizeof (int) );
    results[1] = (int *) malloc ( (k+1) * sizeof (int) );
    results[2] = (int *) malloc ( (n) * sizeof (int) );


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);


    // Function given to implement
    operations ( n, k, m, x, dis, speed, results );


    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    // First crossing vehicles list
    for ( int i=0; i<=k; i++ )  {
        fprintf( outputfilepointer, "%d ", results[0][i]);
    }
    fprintf( outputfilepointer, "\n");


    //Last crossing vehicles list
    for ( int i=0; i<=k; i++ )  {
        fprintf( outputfilepointer, "%d ", results[1][i]);
    }
    fprintf( outputfilepointer, "\n");


    //Total time taken by each vehicles to pass the highway
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", results[2][i]);
    }
    fprintf( outputfilepointer, "\n");

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    return 0;
}