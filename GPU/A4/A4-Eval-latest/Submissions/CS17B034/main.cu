
#include <stdbool.h>
#include<stdio.h>
// #include <thrust/sort.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
using namespace std;
#define ORDER 8
int order = 8;
typedef struct node{
    int num_keys;
    bool is_leaf;
    int* keys;
    struct node* par;
    void** pointers;
} node;
typedef struct record{
    int* val;
}record;
node* currleaf;
int getLeftIndex(node *parent, node *left);
node *ins_leaf(node *leaf, int key, record *pointer);
node *ins_leaf_split(node *root, node *leaf, int key,
                   record *pointer);
node *ins_node_split(node *root, node *parent,
                   int left_index,
                   int key, node *right);
node *ins_par(node *root, node *left, int key, node *right);
node *insert(node *root, int key, int* values,int m);
int height(node* n)
{
    int h = 0;
    node* c = n;
    while(! c->is_leaf){
        c = (node*)c->pointers[0];
        h++;
    }
    return h;
}
__device__ __host__ void printLeaves(node *root, int ord) {
    if (root == NULL) {
        printf("Empty tree.\n");
        return;
    }
    for(int i=0;i<root->num_keys;i++) printf("%d ",root->keys[i]);
    printf("\n");
    int i;
    node *c = root;
    while (!c->is_leaf)
        c =(node*) c->pointers[0];
    while (true) {
        for (i = 0; i < c->num_keys; i++) {
            printf("%d ", c->keys[i]);
        }
        if (c->pointers[ord - 1] != NULL) {
            printf(" | ");
            c = (node*)c->pointers[ord - 1];
        } 
        else break;
    }
    printf("\n");
}
__global__ void pathtrace(node* n,int key, int* out)
{
    node* curr = n;
    int cnt = 0;
    while(! curr-> is_leaf)
    {
        int i = 0;
        while( i < curr->num_keys)
        {
            if(key >= curr->keys[i])
            {
                i++;
            }
            else break;
        }
        out[cnt] = curr->keys[0];
        cnt++;
        curr = (node*)curr->pointers[i];
    }
    out[cnt] =  curr->keys[0];
    cnt++;
}
__global__ void pathtracelen(node* n,int key, int* cnt)
{
    node* curr = n;
    
    while(! curr-> is_leaf)
    {
        int i = 0;
        while( i < curr->num_keys)
        {
            if(key >= curr->keys[i])
            {
                i++;
            }
            else break;
        }
        cnt[0]++;
        curr = (node*)curr->pointers[i];
    }
    cnt[0]++;
}
__global__ void search(node* n,int* keys,int m,int* out,int p)
{
    //key is present or not doesn't matter
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < p){
        int key = keys[id];
        if (n== NULL)
        {
            out[m*id] = -1;
        }
        else{
            node* curr = n;
            
            while(! curr-> is_leaf)
            {
                int i = 0;
                while( i < curr->num_keys)
                {
                    if(key >= curr->keys[i])
                    {
                        i++;
                    }
                    else break;
                }
                curr = (node*)curr->pointers[i];
            }
            bool f = false;
            for(int i=0;i<curr->num_keys;i++)
            {
                if(curr->keys[i]==key)
                {
                    for(int j=0;j<m;j++)
                    {
                        out[m*id+j] = ((record*)curr->pointers[i])->val[j];
                    }
                    f = true;
                    break;
                }
            }
            if(!f)
            {
                out[m*id] = -1;
            }
        }
    }
}
__global__ void findrange(node* n,int* as, int* bs, int m, int*** ans,int p, int order)
{
    int id =  blockIdx.x * blockDim.x + threadIdx.x;
    if(id < p){
        int a = as[id];
        int b = bs[id];
        if (n== NULL)
        {
            ans[id] = NULL;
            return;
        }
        node* curr = n;
        
        while(! curr-> is_leaf)
        {
            int i = 0;
            while( i < curr->num_keys)
            {
                if(a >= curr->keys[i])
                {
                    i++;
                }
                else break;
            }
            curr = (node*)curr->pointers[i];
        }
        int i = 0;
        while(curr !=NULL)
        {
            while(i< curr->num_keys && curr->keys[i]<a) i++;
            if (i== curr->num_keys )
            {
                curr = (node*) curr->pointers[order-1];
                i=0;
            }
            else break;
        }
        if (curr== NULL)
        {
           ans[id] = NULL;
            return;
        }
        int cnt = 0;
        while(curr != NULL)
        {
            while(i < curr->num_keys && curr->keys[i] <= b)
            {
                ans[id][cnt] = ((record*)curr->pointers[i])->val;
                cnt++;
                i++;
            }
            curr = (node*) curr->pointers[order-1];
            i = 0;
        }
        if(cnt==0)
        {
            ans[id] = NULL;
            return;
        }
    }
}
__global__ void findrangelen(node* n,int* as, int* bs, int m, int* ans, int p, int order)
{
    int id =  blockIdx.x * blockDim.x + threadIdx.x;
    if(id < p){
        int a = as[id];
        int b = bs[id];
        if (n== NULL){
            ans[id] = 0;
            return;
        }
        node* curr = n;
        while(! curr-> is_leaf){
            int i = 0;
            while( i < curr->num_keys){
                if(a >= curr->keys[i])
                {
                    i++;
                }
                else break;
            }
            curr = (node*)curr->pointers[i];
        }
        int i = 0;
        while(curr !=NULL){
            while(i< curr->num_keys && curr->keys[i]<a) i++;
            if (i== curr->num_keys ){
                curr = (node*) curr->pointers[order-1];
                i=0;
            }
            else break;
        }
        if (curr== NULL){
           ans[id] = 0;
            return;
        }
        int cnt = 0;
        while(curr != NULL){
            while(i < curr->num_keys && curr->keys[i] <= b){
                cnt++;
                i++;
            }
            curr = (node*) curr->pointers[order-1];
            i = 0;
        }
        
        ans[id] = cnt;
        return;
        
    }
}
__global__ void addition(node* n, int* keys , int* ans, int* values, int p)
{
    int id =  blockIdx.x * blockDim.x + threadIdx.x;
    if (id<p){
        int key = keys[id];
        int an = ans[id];
        int value = values[id]; 
        if (n== NULL)
        {
            return;
        }
        node* curr = n;
        
        while(! curr-> is_leaf)
        {
            int i = 0;
            while( i < curr->num_keys)
            {
                if(key >= curr->keys[i])
                {
                    i++;
                }
                else break;
            }
            curr = (node*)curr->pointers[i];
        }
        for(int i=0;i<curr->num_keys;i++)
        {
            if(curr->keys[i]==key)
            {
                atomicAdd(  &(((record*)curr->pointers[i])->val[an-1]), value);
                return;
            }
        }
    }
    
}
record* mkrecord(int* val, int m)
{
    record* r = (record*) malloc(sizeof(record));
    r->val = (int*) malloc(m*sizeof(int));
    for(int i=0;i<m;i++) r->val[i] = val[i];
    return r;
}
node *mknode(void) {
  node *new_node;
  new_node = (node*)malloc(sizeof(node));
  new_node->keys = (int*)malloc((order - 1) * sizeof(int));
  new_node->pointers = (void**) malloc(order * sizeof(void *));
  new_node->is_leaf = false;
  new_node->num_keys = 0;
  new_node->par = NULL;
  return new_node;
}
node *mkleaf(void) {
  node* leaf = mknode();
  leaf->is_leaf = true;
  return leaf;
}
node *mkroot(int key, int* val, int m) {
  node *root = mkleaf();
  root->keys[0] = key;
  record* r = mkrecord(val,m);
  root->pointers[0] = r;
  root->pointers[order - 1] = NULL;
  root->par = NULL;
  root->num_keys++;
  return root;
}

int getLeftIndex(node *parent, node *left) {
  int left_index = 0;
  while (left_index <= parent->num_keys &&
       parent->pointers[left_index] != left)
    left_index++;
  return left_index;
}

node *ins_leaf_split(node* n,node* l,int key,record* r)
{
    node* nl = mkleaf();
    int* temp_keys = (int*)malloc(order * sizeof(int));
    void** temp_pointers = (void**)malloc(order * sizeof(void *));
    int k = 0;
    while (k < order - 1 && l->keys[k] < key) k++;
    int i,j;
  for (i = 0, j = 0; i < l->num_keys; i++, j++) {
    if (j == k)
      j++;
    temp_keys[j] = l->keys[i];
    temp_pointers[j] = l->pointers[i];
  }

  temp_keys[k] = key;
  temp_pointers[k] = r;

  l->num_keys = 0;
   int split = 0;
  if ((order - 1)%2==0)
  {
      split = (order-1)/2;
  }
  else split = (order-1)/2 + 1;

  for (i = 0; i < split; i++) {
    l->pointers[i] = temp_pointers[i];
    l->keys[i] = temp_keys[i];
    l->num_keys++;
  }

  for (i = split, j = 0; i < order; i++, j++) {
    nl->pointers[j] = temp_pointers[i];
    nl->keys[j] = temp_keys[i];
    nl->num_keys++;
  }

  free(temp_pointers);
  free(temp_keys);

  nl->pointers[order - 1] = l->pointers[order - 1];
  l->pointers[order - 1] = nl;

  for (i = l->num_keys; i < order - 1; i++)
    l->pointers[i] = NULL;
  for (i = nl->num_keys; i < order - 1; i++)
    nl->pointers[i] = NULL;

  nl->par = l->par;

  return ins_par(n, l, nl->keys[0], nl);

}

node *ins_node_split(node *root, node *old_node, int left_index,
                   int key, node *right) {
  int i, j, split;
  node *new_node, *child;
  int *temp_keys;
  node **temp_pointers;

  temp_pointers = (node**) malloc((order + 1) * sizeof(node *));
  temp_keys = (int*)malloc(order * sizeof(int));
  for (i = 0, j = 0; i < old_node->num_keys + 1; i++, j++) {
    if (j == left_index + 1)
      j++;
    temp_pointers[j] = (node*) old_node->pointers[i];
  }

  for (i = 0, j = 0; i < old_node->num_keys; i++, j++) {
    if (j == left_index)
      j++;
    temp_keys[j] = old_node->keys[i];
  }

  temp_pointers[left_index + 1] = right;
  temp_keys[left_index] = key;

  if ((order-1)%2==0)
  {
      split = (order-1)/2;
  }
  else split = (order-1)/2 + 1;

  new_node = mknode();
  old_node->num_keys = 0;
  for (i = 0; i < split; i++) {
    old_node->pointers[i] = temp_pointers[i];
    old_node->keys[i] = temp_keys[i];
    old_node->num_keys++;
  }
  old_node->pointers[i] = temp_pointers[i];
  int spkey = temp_keys[split];
  for (++i, j = 0; i < order; i++, j++) {
    new_node->pointers[j] = temp_pointers[i];
    new_node->keys[j] = temp_keys[i];
    new_node->num_keys++;
  }
  new_node->pointers[j] = temp_pointers[i];

  free(temp_pointers);
  free(temp_keys);

  new_node->par = old_node->par;
  for (i = 0; i <= new_node->num_keys; i++) {
    child = (node*) new_node->pointers[i];
    child->par = new_node;
  }

  return ins_par(root, old_node, spkey, new_node);
}

node *ins_par(node *root, node *left, int key, node *right) {
  node *parent;

  parent = left->par;

  if (parent == NULL){
    node *r = mknode();
    r->keys[0] = key;
    r->pointers[0] = left;
    r->pointers[1] = right;
    r->num_keys++;
    r->par = NULL;
    left->par = r;
    right->par = r;
    return r;
  }

  int li = getLeftIndex(parent, left);

  if (parent->num_keys < order - 1){
      int i;
        for (i = parent->num_keys; i > li; i--) {
            parent->pointers[i + 1] = parent->pointers[i];
            parent->keys[i] = parent->keys[i - 1];
        }
        parent->pointers[li + 1] = right;
        parent->keys[li] = key;
        parent->num_keys++; 
        return root;
   }

  return ins_node_split(root, parent, li, key, right);
}
node *insert(node *n, int key, int* val, int m)
{
    if (n== NULL)
    {
        return mkroot(key, val , m);
    }
        
        node* curr = n;
        while(! curr-> is_leaf)
        {
            int i = 0;
            while( i < curr->num_keys)
            {
                if(key >= curr->keys[i])
                {
                    i++;
                }
                else break;
            }
            curr = (node*)curr->pointers[i];
        }
        for(int i=0;i<curr->num_keys;i++)
        {
            if(curr->keys[i]==key)
            {
                for(int j=0;j<m;j++)
                {
                    ((record*)curr->pointers[i])->val[j] = val[j];
                }
                return n;
            }
        }
        record* r = mkrecord(val,m);
        // curr leaf found
        if(curr->num_keys < order-1)
        {
            // printf("Leaf Insertion\n");
            int j = 0;
            while(j< curr->num_keys && curr->keys[j]<key) j++;
            int num = curr->num_keys;
            for(int i=num;i>j;i--)
            {
                curr->keys[i] = curr->keys[i-1];
                curr->pointers[i] = curr->pointers[i-1];
            }
            curr->keys[j] = key;
            curr->pointers[j] = r;
            curr->num_keys++;
            // return l;
            // curr = ins_leaf(curr,key,r);
            return n;
        }
        // printf("Leaf Split insertion\n");
        return ins_leaf_split(n,curr,key,r);
}
record* copy_rec(record* r,int  m)
{
     record* gr;
    hipMalloc(&gr, sizeof(record) ) ;
         int* gval;
        hipMalloc(&(gval) , m*sizeof(int) );
        record* gt;
        gt = (record*)malloc(sizeof(record));
        hipMemcpy(gval,r->val,m*sizeof(int),hipMemcpyHostToDevice);
     gt->val = gval;
    hipMemcpy(gr,gt,sizeof(record),hipMemcpyHostToDevice);
    return gr;
}
node* copyleaf(node* root,node* par,int m)
{
    if (root==NULL) return NULL;
     node* gtree = mknode();
    // int num_keys;
    // bool is_leaf;
    // int* keys;
    // struct node* par;
    // void** pointers;
    // cudaMemcpy(gtree->num_keys,root->num_keys,sizeof(int),cudaMemcpyHosttoDevice);
    // cudaMemcpy(gtree->is_leaf,root->is_leaf,sizeof(bool),cudaMemcpyHosttoDevice);
    int* keys;
    void** pointers;
    hipMalloc(&keys, (order-1)*sizeof(int));
    hipMalloc(&pointers, (order)*sizeof(void*));
    hipMemcpy(keys,root->keys,(order-1)*sizeof(int),hipMemcpyHostToDevice);
    gtree->keys = keys;
    gtree->num_keys = root->num_keys;
    gtree->is_leaf = root->is_leaf;
    gtree->par = par;
    node* gputree;
    hipMalloc(&gputree,sizeof(node));
    void** gpointers;
    gpointers = (void**) malloc((order)*sizeof(void*));
    for(int i=0;i<root->num_keys;i++)
    {
            gpointers[i] = copy_rec((record*)root->pointers[i],m);
    } 
    for(int i=root->num_keys;i<order;i++) gpointers[i] = NULL;
    hipMemcpy(pointers,gpointers,(order)*sizeof(void*),hipMemcpyHostToDevice);
    gtree->pointers = pointers;
    hipMemcpy(gputree,gtree,sizeof(node),hipMemcpyHostToDevice);
    return gputree;
}
__global__ void helper(node* l1,node* lnext,int order)
{
    l1->pointers[order-1] = lnext;
}
node *copytree(node* root, node* par,int m)
{
    // printf("copying %d node\n",root->keys[0]);
    if (root == NULL) return NULL;
    if(root->is_leaf)
    {
        node* l =  copyleaf(root,par,m);
        if(currleaf!=NULL){
            helper<<<1,1>>>(currleaf,l,order);
            hipDeviceSynchronize();
        }
        currleaf = l;
        return l;
    }
    node* gtree = mknode();
    // int num_keys;
    // bool is_leaf;
    // int* keys;
    // struct node* par;
    // void** pointers;
    // cudaMemcpy(gtree->num_keys,root->num_keys,sizeof(int),cudaMemcpyHosttoDevice);
    // cudaMemcpy(gtree->is_leaf,root->is_leaf,sizeof(bool),cudaMemcpyHosttoDevice);
    int* keys;
    void** pointers;
    hipMalloc(&keys, (order-1)*sizeof(int));
    hipMalloc(&pointers, (order)*sizeof(void*));
    hipMemcpy(keys,root->keys,(order-1)*sizeof(int),hipMemcpyHostToDevice);
    gtree->keys = keys;
    gtree->num_keys = root->num_keys;
    gtree->is_leaf = root->is_leaf;
    gtree->par = par;
    node* gputree;
    hipMalloc(&gputree,sizeof(node));
    void** gpointers;
    gpointers = (void**) malloc((order)*sizeof(void*));
    for(int i=0;i<root->num_keys + 1;i++)
    {
            node* cpnode = copytree((node*)root->pointers[i],gputree,m);
            gpointers[i] = cpnode;
    }
    hipMemcpy(pointers,gpointers,(order)*sizeof(void*),hipMemcpyHostToDevice);
    gtree->pointers = pointers;
    hipMemcpy(gputree,gtree,sizeof(node),hipMemcpyHostToDevice);
    return gputree;
}
__global__ void printGPULeaves(node* root,int order){
    // printf("Entered GPU\n");
    printLeaves(root,order);
}
__global__ void printk(int* val,int m)
{
    for(int i=0;i<m;i++) printf("%d ",val[i]);
    printf("\n");
}
int main(int argc,char **argv){

    //variable declarations
    int n,m;
    
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen( outputfilename , "w");
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0;
    }
    if( outputfilepointer == NULL) {
        printf(" output.txt file failed to open. ");
        return 0;
    }
    
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of vehicles
    fscanf( inputfilepointer, "%d", &m );      //scaning for number of toll tax zones
    int val[n][m];
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            fscanf( inputfilepointer, "%d", &val[i][j] );
        }
    }
    node* root;
    node* groot;
    root = NULL;
    groot = NULL;
    for(int i=0;i<n;i++){
        root = insert(root,val[i][0],val[i],m);
    }
    currleaf = NULL;
    groot = copytree(root,NULL,m);
    
    // printLeaves(root,order);
    // printf("GPU Leaves\n");
    // printGPULeaves<<<1,1>>>(groot,order);
    // cudaDeviceSynchronize();
    
    int q;
    fscanf( inputfilepointer, "%d", &q );
    for(int i=0;i<q;i++)
    {
            int op;
            fscanf(inputfilepointer,"%d",&op);
            // printf("operation = %d\n",op);
            if(op==1)
            {
                int num_op;
                fscanf(inputfilepointer,"%d",&num_op);
                int* keys = (int*) malloc(num_op*sizeof(int));
                int* gkeys;
                hipMalloc(&gkeys,num_op*sizeof(int));
                int* out;
                hipMalloc(&out,num_op*m*sizeof(int));
                int* pout;
                pout = (int*)malloc(num_op*m*sizeof(int));
                for(int j=0;j<num_op;j++)
                {
                    int k;
                    fscanf(inputfilepointer,"%d",&k);
                    keys[j] = k;
                }
                hipMemcpy(gkeys,keys,num_op*sizeof(int),hipMemcpyHostToDevice);
                int a = num_op/1024;
                a+=1;
                search<<<a,1024>>>(groot,gkeys,m,out,num_op);
                
                hipDeviceSynchronize();
                hipMemcpy(pout,out,num_op*m*sizeof(int),hipMemcpyDeviceToHost);
                for(int j=0;j<num_op;j++){
                    if(pout[j*m]!=-1){
                        for(int k=0;k<m;k++){
                            fprintf(outputfilepointer, "%d ",pout[j*m+k]);
                        }
                        fprintf(outputfilepointer,"\n");
                    }
                    else fprintf(outputfilepointer,"-1\n");
                }
            }
            if(op==2)
            {
                int num_op;
                fscanf(inputfilepointer,"%d",&num_op);
                int* as = (int*) malloc(num_op*sizeof(int));
                int* gas;
                hipMalloc(&gas,num_op*sizeof(int));
                int* bs = (int*) malloc(num_op*sizeof(int));
                int* gbs;
                hipMalloc(&gbs,num_op*sizeof(int));
                for(int j=0;j<num_op;j++)
                {
                    int a,b;
                    fscanf(inputfilepointer,"%d",&a);
                    fscanf(inputfilepointer,"%d",&b);
                    as[j] = a;
                    bs[j] = b;
                }
                int a = num_op/1024;
                a+=1;
                hipMemcpy(gas,as,num_op*sizeof(int),hipMemcpyHostToDevice);
                hipMemcpy(gbs,bs,num_op*sizeof(int),hipMemcpyHostToDevice);
                 int* lens = (int*) malloc(num_op*sizeof(int));
                int* glens;
                hipMalloc(&glens,num_op*sizeof(int));
                findrangelen<<<a,1024>>>(groot,gas,gbs,m,glens,num_op,order);
                hipDeviceSynchronize();
                hipMemcpy(lens,glens,num_op*sizeof(int),hipMemcpyDeviceToHost);
                int*** recs = (int***) malloc(num_op*sizeof(int**));
                int*** grecs;
                hipMalloc(&grecs,num_op*sizeof(int**));
                for(int j=0;j<num_op;j++)
                {
                    hipMalloc(&recs[j],lens[j]*sizeof(int*));
                }
                hipMemcpy(grecs,recs,num_op*sizeof(int**),hipMemcpyHostToDevice);
                findrange<<<a,1024>>>(groot,gas,gbs,m,grecs,num_op,order);
                hipDeviceSynchronize();

                 for(int j=0;j<num_op;j++)
                {
                    if(lens[j]==0)
                    {
                        fprintf(outputfilepointer,"-1\n");
                        continue;
                    }
                    // printf("%d\n",j);
                    int** rec = (int**) malloc(lens[j]*sizeof(int*));
                    hipMemcpy(rec,recs[j],lens[j]*sizeof(int*),hipMemcpyDeviceToHost);
                    // printk<<<1,1>>>(rec[0],m);
                    // cpkernel<<<1,1>>>(gvals,grecs,j,lens[j]);
                    // cudaDeviceSynchronize();
                    // cudaMemcpy(vals,rec[k],m*sizeof(int),cudaMemcpyDeviceToHost);
                    for(int k=0;k<lens[j];k++)
                    {
                        int* vals = (int*) malloc(m*sizeof(int));
                        hipMemcpy(vals,rec[k],m*sizeof(int),hipMemcpyDeviceToHost);
                        for(int l=0;l<m;l++)
                        {
                             fprintf(outputfilepointer, "%d ", vals[l]);
                        }
                        fprintf(outputfilepointer,"\n");
                        free(vals);
                    }
                    free(rec);
                }
            }
            if(op==3)
            {
                int num_op;
                fscanf(inputfilepointer,"%d",&num_op);
                int* keys = (int*) malloc(num_op*sizeof(int));
                int* gkeys;
                hipMalloc(&gkeys,num_op*sizeof(int));
                int* ans = (int*) malloc(num_op*sizeof(int));
                int* gans;
                hipMalloc(&gans,num_op*sizeof(int));
                int* incs = (int*) malloc(num_op*sizeof(int));
                int* gincs;
                hipMalloc(&gincs,num_op*sizeof(int));
                for(int j=0;j<num_op;j++)
                {
                    int k,an,inc;
                    fscanf(inputfilepointer,"%d",&k);
                    fscanf(inputfilepointer,"%d",&an);
                    fscanf(inputfilepointer,"%d",&inc);
                    keys[j] = k;
                    ans[j] = an;
                    incs[j] = inc;
                }
                hipMemcpy(gkeys,keys,num_op*sizeof(int),hipMemcpyHostToDevice);
                hipMemcpy(gans,ans,num_op*sizeof(int),hipMemcpyHostToDevice);
                hipMemcpy(gincs,incs,num_op*sizeof(int),hipMemcpyHostToDevice);
                int a = num_op/1024;
                a+=1;
                addition<<<a,1024>>>(groot,gkeys,gans,gincs,num_op);
                hipDeviceSynchronize();
            }
            if(op==4)
            {
                int k;
                fscanf(inputfilepointer,"%d",&k);
                int* len =(int*) malloc(sizeof(int));
                len[0] = 0;
                int* glen;
                hipMalloc(&glen,sizeof(int));
                hipMemcpy(glen,len,sizeof(int),hipMemcpyHostToDevice);
                pathtracelen<<<1,1>>>(groot,k,glen);
                hipDeviceSynchronize();
                hipMemcpy(len,glen,sizeof(int),hipMemcpyDeviceToHost);
                int* gout;
                int* out = (int*) malloc(len[0]*sizeof(int));
                hipMalloc(&gout,len[0]*sizeof(int));
                pathtrace<<<1,1>>>(groot,k,gout);
                hipDeviceSynchronize();
                hipMemcpy(out,gout,len[0]*sizeof(int),hipMemcpyDeviceToHost);
                for(int i=0;i<len[0];i++)
                {
                    fprintf(outputfilepointer,"%d ",out[i]);
                }
                fprintf(outputfilepointer,"\n");
            }
            
    }
    fclose( outputfilepointer );
    fclose( inputfilepointer );
    return 0;
}