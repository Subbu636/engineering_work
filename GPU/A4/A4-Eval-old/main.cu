#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <bits/stdc++.h>

#include <sys/time.h> 

#define fanout 7

struct bp_node{
    bool leaf;
    int  key[fanout],loc[fanout],len; // loc varible for leaf points to index of 2d array
    bp_node *next[fanout+1],*parent;
};

__host__ __device__ void search_trace(bp_node *root,int k, int *trace){
    trace[0] = -1;
    if(!root) return;
    int c = 0;
    bp_node *ptr = root;
    trace[c] = ptr->key[0];
    c++;
    while(!ptr->leaf){
        for(int i = 0;i < ptr->len;i++){
            if(k < ptr->key[i]){
                ptr = ptr->next[i];
                trace[c] = ptr->key[0];
                c++;
                break;
            }
            if (i == ptr->len - 1) {
                ptr = ptr->next[i + 1];
                trace[c] = ptr->key[0];
                c++;
                break;
            }
        }
    }
    trace[c] = -1;
    return;
}

__host__ __device__ int search_loc(bp_node *root,int k){
    if(!root) return -1;
    bp_node *ptr = root;
    while(!ptr->leaf){
        for(int i = 0;i < ptr->len;i++){
            if(k < ptr->key[i]){
                ptr = ptr->next[i];
                break;
            }
            if (i == ptr->len - 1) {
                ptr = ptr->next[i + 1];
                break;
            }
        }
    }
    for(int i = 0;i < ptr->len;i++){
        if(ptr->key[i] == k){
            return ptr->loc[i];
        }
    }
    return -1;
}

__host__ __device__ bp_node* search_lnode(bp_node *root,int k){
    if(!root) return NULL;
    bp_node *ptr = root;
    while(!ptr->leaf){
        for(int i = 0;i < ptr->len;i++){
            if(k < ptr->key[i]){
                ptr = ptr->next[i];
                break;
            }
            if (i == ptr->len - 1) {
                ptr = ptr->next[i + 1];
                break;
            }
        }
    }
    return ptr;
}

__host__ __device__ void print_dfs(bp_node *root){
    if(!root) return;
    printf("%d/%d :",root->len,(int)root->leaf);
    for(int i = 0;i < root->len;++i){
        printf(" %d",root->key[i]);
    }
    printf("\n");
    if(root->leaf) return;
    for(int i = 0;i < root->len+1;++i){
        print_dfs(root->next[i]);
    }
}

bp_node* insert_inode(int k, bp_node* inode, bp_node* cnode, bp_node* root){
	if(inode->len < fanout){
        int p = 0;
        for(int i = 0; i < inode->len;++i){
            if(k > inode->key[i]){
                p++;
                continue;
            }
            int v = inode->len + p - i;
            inode->key[v] = inode->key[v-1];
            inode->next[v+1] = inode->next[v];
        }
		inode->key[p] = k;
		inode->len++;
		inode->next[p + 1] = cnode;
        return root;
	}
    int ktemp[fanout+1];
    bp_node* ntemp[fanout+2];
    std::copy(inode->key,(inode->key)+fanout,ktemp);
    std::copy(inode->next,(inode->next)+fanout+1,ntemp);
    int p = 0;
    for(int i = 0;i < fanout;i++){
        if(k > ktemp[i]){
            p++;
            continue;
        }
        int v = fanout + p - i;
        ktemp[v] = ktemp[v-1];
        ntemp[v+1] = ntemp[v];
    }
    ktemp[p] = k;
    ntemp[p+1] = cnode;

    bp_node *ninode = (bp_node*)malloc(sizeof(bp_node));
    ninode->leaf = false;
    inode->len = ceil((float)fanout/2.0);
    ninode->len = fanout - inode->len;
    std::copy(ktemp,ktemp+inode->len,inode->key);
    std::copy(ntemp,ntemp+inode->len+1,inode->next);
    std::copy(ktemp+inode->len+1,ktemp+fanout+1,ninode->key);
    std::copy(ntemp+inode->len+1,ntemp+fanout+2,ninode->next);
    ninode->parent = inode->parent;
    for(int i = 0;i < ninode->len+1;i++){
        ninode->next[i]->parent = ninode;
    }
    for(int i = 0;i < inode->len+1;++i){
        inode->next[i]->parent = inode;
    }
    if (inode == root) {
        bp_node *nroot = (bp_node*)malloc(sizeof(bp_node));
        nroot->key[0] = ktemp[inode->len];
        nroot->next[0] = inode;
        nroot->next[1] = ninode;
        nroot->leaf = false;
        nroot->len = 1;
        inode->parent = nroot;
        ninode->parent = nroot;
        return nroot;
    }
    return insert_inode(ktemp[inode->len],inode->parent,ninode,root);
}

bp_node* insert_key(bp_node *root,int k,int l){
    if (!root) {
        bp_node *root = (bp_node*)malloc(sizeof(bp_node));
        root->key[0] = k;
        root->loc[0] = l;
        root->leaf = true;
        root->len = 1;
        return root;
    }
    bp_node *ptr = root;
    while(!ptr->leaf){
        for(int i = 0;i < ptr->len;i++){
            if(k < ptr->key[i]){
                ptr = ptr->next[i];
                break;
            }
            if (i == ptr->len - 1) {
                ptr = ptr->next[i + 1];
                break;
            }
        }
    }
    if(ptr->len < fanout){
        int p = 0;
        for(int i = 0;i < ptr->len;i++){
            if(k > ptr->key[i]){
                p++;
                continue;
            }
            int v = ptr->len + p - i;
            ptr->key[v] = ptr->key[v-1];
            ptr->loc[v] = ptr->loc[v-1];
        }
        ptr->len++;
        ptr->key[p] = k;
        ptr->loc[p] = l;
        ptr->next[ptr->len] = ptr->next[ptr->len - 1];
        ptr->next[ptr->len - 1] = NULL;
        return root;
    }
    int ktemp[fanout+1], ltemp[fanout+1];
    std::copy(ptr->key,(ptr->key)+fanout,ktemp);
    std::copy(ptr->loc,(ptr->loc)+fanout,ltemp);
    int p = 0;
    for(int i = 0;i < fanout;i++){
        if(k > ktemp[i]){
            p++;
            continue;
        }
        int v = fanout + p - i;
        ktemp[v] = ktemp[v-1];
        ltemp[v] = ltemp[v-1];
    }
    ktemp[p] = k;
    ltemp[p] = l;

    bp_node *nleaf = (bp_node*)malloc(sizeof(bp_node));
    nleaf->leaf = true;
    ptr->len = ceil((float)fanout/2.0);
    nleaf->len = fanout + 1 - ptr->len;
    ptr->next[ptr->len] = nleaf;
    nleaf->next[nleaf->len] = ptr->next[fanout];
    ptr->next[fanout] = NULL;
    nleaf->parent = ptr->parent;
    std::copy(ktemp,ktemp+ptr->len,ptr->key);
    std::copy(ltemp,ltemp+ptr->len,ptr->loc);
    std::copy(ktemp+ptr->len,ktemp+fanout+1,nleaf->key);
    std::copy(ltemp+ptr->len,ltemp+fanout+1,nleaf->loc);
    
    if(ptr == root){
        bp_node *nroot = (bp_node*)malloc(sizeof(bp_node));
        nroot->key[0] = nleaf->key[0];
        nroot->next[0] = ptr;
        nroot->next[1] = nleaf;
        nroot->leaf = false;
        nroot->len = 1;
        ptr->parent = nroot;
        nleaf->parent = nroot;
        return nroot;
    }
    return insert_inode(nleaf->key[0],ptr->parent,nleaf,root);
}

void bpt_cudamem(bp_node *croot, bp_node *groot,bp_node **lfs,bp_node **cfrom,bp_node **cto,int *cnum,int *c){
    if(!croot) return;
    hipMemcpy(&groot->leaf,&croot->leaf,sizeof(bool),hipMemcpyHostToDevice);
    hipMemcpy(&groot->len,&croot->len,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(groot->key,croot->key,fanout*sizeof(int),hipMemcpyHostToDevice);
    if(croot->leaf){
        hipMemcpy(groot->loc,croot->loc,fanout*sizeof(int),hipMemcpyHostToDevice);
        lfs[c[0]] = groot;
        c[0]++;
    }
    else{
        for(int i = 0;i < croot->len+1;++i){
            bp_node *gnode;
            hipMalloc(&gnode,sizeof(bp_node));
            cfrom[c[1]] = groot;
            cnum[c[1]] = i;
            cto[c[1]] = gnode;
            c[1]++;
            bpt_cudamem(croot->next[i],gnode,lfs,cfrom,cto,cnum,c);
        }
    }

}

__global__ void just_join_lfs(bp_node **glfs, int c){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= c-1) return;
    glfs[id]->next[glfs[id]->len] = glfs[id+1];
    // printf("%d",glfs[id]->key[0]);
}

__global__ void just_join_inodes(bp_node **gfrom,int *gnum,bp_node **gto, int c){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= c) return;
    gfrom[id]->next[gnum[id]] = gto[id];
}

__global__ void just_print(bp_node *groot){
    print_dfs(groot);
}

__global__ void cuda_search(int *garr,bp_node *groot,int p){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= p) return;
    garr[id] = search_loc(groot,garr[id]);
}

__global__ void cuda_range_len(int *garr,int *glen,bp_node **gptr,bp_node *groot,int p){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= p) return;
    bp_node *ptr = search_lnode(groot,garr[id*2]);
    gptr[id] = ptr;
    glen[id] = 0;
    while(ptr){
        // printf("In:%d.%d.%d\n",ptr->key[0],garr[id*2],garr[id*2+1]);
        if(ptr->key[0] > garr[2*id+1]) break;
        for(int i = 0;i < ptr->len;++i){
            if(ptr->key[i] >= garr[2*id] && ptr->key[i] <= garr[2*id+1]) glen[id]++;
        }
        ptr = ptr->next[ptr->len];
    }
    // printf("len:%d,%d\n",id,glen[id]);
}

__global__ void cuda_range_val(int *garr,bp_node **gptr,int **gans,int p){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= p) return;
    bp_node *ptr = gptr[id];
    int c = 0;
    // printf("ptr1:%p\n",ptr);
    while(ptr){
        if(ptr->key[0] > garr[2*id+1]) break;
        for(int i = 0;i < ptr->len;++i){
            if(ptr->key[i] >= garr[2*id] && ptr->key[i] <= garr[2*id+1]){
                gans[id][c] = ptr->loc[i];
                // printf("ptr:%d\n",ptr->loc[i]);
                c++;
            }
        }
        ptr = ptr->next[ptr->len];
    }
    // printf("done:%d\n",id);
}

__global__ void cuda_update(int *garr,int *gloc,int **ggloc,int *gpu_db,bp_node *groot,int p,int m){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= p) return;
    int l = search_loc(groot,garr[3*id]);
    if(l == -1) return;
    atomicAdd(&gpu_db[l*m + garr[3*id+1]-1],garr[3*id+2]);
    gloc[id] = l*m + garr[3*id+1]-1;
    ggloc[id] = &gpu_db[l*m + garr[3*id+1]-1];
}

int main(int argc,char **argv){
    hipDeviceSetLimit(hipLimitStackSize, 1 << 25);

    // timings
    struct timeval t1, t2;
    std::vector <double> kerneltime;

    // input reading
    FILE *inp;
    char *fname = argv[1];
    inp = fopen(fname,"r");
    if (!inp)  {
        printf("#cannot open input file");
        return 0;
    }
    bp_node *root = NULL;
    int n,m,q;
    fscanf(inp,"%d",&n);
    fscanf(inp,"%d",&m);
    int *db = (int*) malloc(n*m*sizeof(int));
    for(int i = 0;i < n;i++){
        int v;
        fscanf(inp,"%d",&v);
        root = insert_key(root,v,i);
        db[i*m] = v;
        for(int j = 1;j < m;++j){
            fscanf(inp,"%d",&db[i*m+j]);
        }
    }

    // allocate and copy gpu mem
    bp_node *groot;
    hipMalloc(&groot,sizeof(bp_node));
    bp_node **lfs = (bp_node**)malloc(n*sizeof(bp_node*));
    bp_node **cfrom = (bp_node**)malloc(2*n*sizeof(bp_node*));
    bp_node **cto = (bp_node**)malloc(2*n*sizeof(bp_node*));
    int *cnum = (int*) malloc(2*n*sizeof(int));
    int *c = (int*) malloc(2*sizeof(int));
    c[0] = 0;c[1] = 0;
    bpt_cudamem(root,groot,lfs,cfrom,cto,cnum,c);
    hipDeviceSynchronize();
    bp_node **glfs;
    hipMalloc(&glfs,c[0]*sizeof(bp_node*));
    hipMemcpy(glfs,lfs,c[0]*sizeof(bp_node*),hipMemcpyHostToDevice);
    just_join_lfs<<<n,1>>>(glfs,c[0]);
    bp_node **gfrom,**gto;
    int *gnum;
    hipMalloc(&gnum,c[1]*sizeof(int));
    hipMemcpy(gnum,cnum,c[1]*sizeof(int),hipMemcpyHostToDevice);
    hipMalloc(&gfrom,c[1]*sizeof(bp_node*));
    hipMemcpy(gfrom,cfrom,c[1]*sizeof(bp_node*),hipMemcpyHostToDevice);
    hipMalloc(&gto,c[1]*sizeof(bp_node*));
    hipMemcpy(gto,cto,c[1]*sizeof(bp_node*),hipMemcpyHostToDevice);
    just_join_inodes<<<2*n,1>>>(gfrom,gnum,gto,c[1]);
    hipDeviceSynchronize();
    int *gpu_db;
    hipMalloc(&gpu_db,n*m*sizeof(int));
    hipMemcpy(gpu_db,db,n*m*sizeof(int),hipMemcpyHostToDevice);

    // output file 
    char *ofname = argv[2]; 
    FILE *op;
    op = fopen(ofname,"w");

    // print_dfs(root);

    // just_print<<<1,1>>>(groot);

    // scan and implement 
    fscanf(inp,"%d",&q);
    for(int l = 0;l < q;++l){
        int t;
        fscanf(inp,"%d",&t);
        if(t == 1){
            int p;
            fscanf(inp,"%d",&p);
            int *arr = (int*) malloc(p*sizeof(int));
            int *garr;
            hipMalloc(&garr,p*sizeof(int));
            for(int i = 0;i < p;++i){
                fscanf(inp,"%d",&arr[i]);
            }
            hipMemcpy(garr,arr,p*sizeof(int),hipMemcpyHostToDevice);
            gettimeofday(&t1, 0);
            cuda_search<<<p,1>>>(garr,groot,p);
            hipDeviceSynchronize();
            gettimeofday(&t2, 0);
            hipMemcpy(arr,garr,p*sizeof(int),hipMemcpyDeviceToHost);
            for(int i = 0;i < p;++i){
                if(arr[i] == -1){
                    fprintf(op,"-1\n");
                    continue;
                }
                for(int j = 0;j < m;++j){
                    fprintf(op,"%d ",db[arr[i]*m+j]);
                }
                fprintf(op,"\n");
            }
        }
        else if(t == 2){
            int p;
            fscanf(inp,"%d",&p);
            int *arr = (int*) malloc(2*p*sizeof(int));
            int *garr,*glen,**gans;
            bp_node **gptr;
            hipMalloc(&garr,2*p*sizeof(int));
            hipMalloc(&glen,p*sizeof(int));
            hipMalloc(&gans,p*sizeof(int*));
            hipMalloc(&gptr,p*sizeof(bp_node*));
            int **ans = (int**)malloc(p*sizeof(int*)), **gloc = (int**)malloc(p*sizeof(int*));
            for(int i = 0;i < 2*p;++i){
                fscanf(inp,"%d",&arr[i]);
            }
            hipMemcpy(garr,arr,2*p*sizeof(int),hipMemcpyHostToDevice);
            cuda_range_len<<<p,1>>>(garr,glen,gptr,groot,p);
            hipDeviceSynchronize();
            int *len = (int*)malloc(p*sizeof(int));
            hipMemcpy(len,glen,p*sizeof(int),hipMemcpyDeviceToHost);
            for(int i = 0;i < p;++i){
                hipMalloc(&gloc[i],len[i]*sizeof(int));
            }
            hipMemcpy(gans,gloc,p*sizeof(int*),hipMemcpyHostToDevice);
            gettimeofday(&t1, 0);
            cuda_range_val<<<p,1>>>(garr,gptr,gans,p);
            hipDeviceSynchronize();
            gettimeofday(&t2, 0);
            // printf("%p,%p,%p,%p,%p\n",glen,gloc[0],garr,len,arr);
            for(int i = 0;i < p;++i){
                if(len[i] == 0){
                    fprintf(op,"-1\n");
                    continue;
                }
                ans[i] = (int*)malloc(len[i]*sizeof(int));
                hipMemcpy(ans[i],gloc[i],len[i]*sizeof(int),hipMemcpyDeviceToHost);
                for(int j = 0;j < len[i];++j){
                    for(int k = 0;k < m;++k){
                        fprintf(op,"%d ",db[ans[i][j]*m+k]);
                    }
                    fprintf(op,"\n");
                }
            }
        }
        else if(t == 3){
            int p;
            fscanf(inp,"%d",&p);
            int *arr = (int*) malloc(3*p*sizeof(int));
            for(int i = 0;i < 3*p;++i){
                fscanf(inp,"%d",&arr[i]);
            }
            int *garr,*gloc,**ggloc,*loc = (int*) malloc(p*sizeof(int)),**cgloc = (int**) malloc(p*sizeof(int*));
            hipMalloc(&garr,3*p*sizeof(int));
            hipMalloc(&gloc,p*sizeof(int));
            hipMalloc(&ggloc,p*sizeof(int*));
            hipMemcpy(garr,arr,3*p*sizeof(int),hipMemcpyHostToDevice);
            gettimeofday(&t1, 0);
            cuda_update<<<p,1>>>(garr,gloc,ggloc,gpu_db,groot,p,m);
            hipDeviceSynchronize();
            gettimeofday(&t2, 0);
            hipMemcpy(loc,gloc,p*sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(cgloc,ggloc,p*sizeof(int*),hipMemcpyDeviceToHost);
            for(int i = 0;i < p;++i){
                hipMemcpy(&db[loc[i]],cgloc[i],sizeof(int),hipMemcpyDeviceToHost);
            }
        }
        else if(t == 4){
            int trace[25] = {0};
            int k;
            fscanf(inp,"%d",&k);
            gettimeofday(&t1, 0);
            search_trace(root,k,trace);
            hipDeviceSynchronize();
            gettimeofday(&t2, 0);
            for(int i = 0;i < 25 && trace[i] != -1;++i){
                fprintf(op,"%d ", trace[i]);
            }
            fprintf(op,"\n");
        }
        double td = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0; // Time taken by kernel in seconds 
		kerneltime.push_back(td);  
        printf("Time taken by kernel to execute is: %.6f ms\n", td);
    }
    int nall = kerneltime.size();
	double sumtime=0;
	for(int i=0;i<nall;i++){
		sumtime += kerneltime[i];
	}
	// print the time taken by all the kernels of the current test-case
    printf("total time taken by the current test-case is %.6f ms\n",sumtime);
    return 0;

    // debugging b+ tree
    // bp_node *root = insert_key(NULL,6,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,16,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,26,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,36,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,46,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,56,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,27,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,28,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,29,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,30,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,31,0);
    // print_dfs(root);
    // printf("--------------------\n");
    // root = insert_key(root,32,0);
    // cudaDeviceSetLimit(cudaLimitStackSize, 1 << 16); // might need be reduced to run on lower end systems
    // print_dfs(root);
    // printf("GPU\n");
    // bp_node *groot;
    // cudaMalloc(&groot,sizeof(bp_node));
    // bpt_cudamem(root,groot);
    // printf("memCopied\n");
    // just_print<<<1,1>>>(groot);
    // cudaDeviceSynchronize();
    // return 0;
}