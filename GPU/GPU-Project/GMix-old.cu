#include "hip/hip_runtime.h"
#include "util.h"

__host__ __device__ double *create(int m, int n){
    return (double*)malloc(sizeof(double)*n*m);
}

__host__ __device__ void init(double *v, int m, int n, double val){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            v[i*n+j] = val;
        }
    }
}

__host__ __device__ void copy(double *a, double *b, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            a[i*m+j] = b[i*m+j];
        }
    }
}


__host__ __device__ double *transpose(double *trans, double *v, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            trans[j*m+i] = v[i*n+j];
        }
    }
    return trans;
}

__host__ __device__ double *matsub(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]-v2[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matmul(double *res, double *a, double *b, int m, int s, int n){
    init(res, m, n, 0.0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < s; k++) res[i*n+j] += a[i*s+k]*b[k*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matadd(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]+v2[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matsmul(double *res, double *v, double val, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = val*v[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *cofactor(double *res, double *a, int p, int q, int n)
{
    int r = 0, c = 0;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i != p && j != q)
            {
                res[r*(n-1)+(c++)] = a[i*n+j];
                if (c == n - 1)
                {
                    c = 0;
                    r++;
                }
            }
        }
    }
    return res;
}

__host__ __device__ double matdet(double *a, int n)
{
    double res = 0.0;
    if (n == 1) return a[0];
    double s = 1.0, *temp = create(n-1,n-1);
    for (int i = 0; i < n; i++)
    {
        res += (s*a[i]*matdet(cofactor(temp,a,0,i,n),n-1));
        s = -1.0*s;
    }
    free(temp);
    return res;
}

__host__ __device__ double *matadj(double *res, double *a, int n)
{
    if(n == 1){
        res[0] = 1.0;
        return res;
    }
    double s = 1.0, *temp = create(n,n);
    for(int i = 0;i < n;i++)
    {
        for(int j = 0;j < n;++j)
        {
            if((i+j)%2 == 0) s = 1.0;
            else s = -1.0;
            res[j*n+i] = (s)*(matdet(cofactor(temp,a,i,j,n), n-1));
        }
    }
    free(temp);
    return res;
}

__host__ __device__ double *matinv(double *res, double *a, int n)
{
    double d = matdet(a,n);
    if (d == 0.0)
    {
        printf("#singular - inverse not possible\n");
        assert(false);
    }
    double *adj = create(n,n);
    matadj(adj,a,n);
    for(int i = 0;i < n;++i){
        for(int j = 0;j < n;++j){
            res[i*n+j] = adj[i*n+j]/d;
        }
    }
    free(adj);
    return res;
}

__host__ __device__ double norm(double *x, double *mu, double *sigma, int d){
    double deno = sqrt(pow((double)(2.0*3.14159),(double)d)*matdet(sigma,d));
    double *dif = create(d,1),*temp = create(d,1),*inv = create(d,d),val[1],*trans = create(d,1);
    matsub(dif,x,mu,d,1);
    double expo = (-0.5)*(matmul(val,matmul(temp,transpose(trans,dif,d,1),matinv(inv,sigma,d),1,d,d),dif,1,d,1)[0]);
    // printf("%f %f\n",matdet(sigma,d),inv[0]);
    // printf("%lf %lf %lf\n",exp(-0.5*expo),deno, expo);
    free(dif); free(temp); free(inv); free(trans);
    return exp(expo)/deno;
}

__host__ __device__ void matprint(double *a, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            printf("%f ",a[i*n+j]);
        }
        printf ("\n");
    }
}


void gmix_cpu(double *p,double *r, int k, int iter, int l, int d, double *ctime){

    // Debugging
    // double arr[4] = {2.0,1.0,1.0,2.0};
    // double mu[2] = {1.0, 1.0}, x[2] = {1.0, 0.0};
    // cout<<norm(x,mu,arr,2)<<endl;

    double n[k],pi[k],mu[k*d],sigma[k*d*d];
    double temp[d], var[d*d],trans[d];
    struct timeval t1, t2;

    matprint(r,2,k);
    cout<<"r----------------------------"<<endl;

    for(int t = 0;t < iter;++t){
        gettimeofday(&t1, 0);
        // M-step
        init(n,1,k,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                n[i] += r[j*k+i];
            }
        }
        // matprint(n,1,k);
        // cout<<"n--------------------------------"<<endl;
        for(int i = 0;i < k;++i){
            pi[i] = n[i]/(double)l;
        }
        // matprint(pi,1,k);
        // cout<<"pi--------------------------------"<<endl;
        init(mu,k,d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matadd(&mu[i*d],&mu[i*d],matsmul(temp,&p[j*d],r[j*k+i],d,1),d,1);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&mu[i*d],&mu[i*d],1/n[i],d,1);
        }
        // matprint(mu,1,k*d);
        // cout<<"mu--------------------------------"<<endl;
        init(sigma,k,d*d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matsub(temp,&p[j*d],&mu[i*d],d,1);
                matmul(var,temp,transpose(trans,temp,d,1),d,1,d);
                matsmul(var,var,r[j*k+i],d,d);
                matadd(&sigma[i*d*d],&sigma[i*d*d],var,d,d);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&sigma[i*d*d],&sigma[i*d*d],1/n[i],d,d);
        }
        // matprint(sigma,1,k*d*d);
        // cout<<"sigma--------------------------------"<<endl;
        gettimeofday(&t2, 0);
        ctime[t*2] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
        cout<<"CPU:"<<t<<endl;
        gettimeofday(&t1, 0);
        // E-step
        for(int i = 0;i < l; ++i){
            double s = 0;
            for(int j = 0;j < k; ++j){
                r[i*k+j] = pi[j]*norm(&p[i*d],&mu[j*d],&sigma[j*d*d],d);
                s+=r[i*k+j];
            }
            for(int j = 0;j < k;++j){
                r[i*k+j] = r[i*k+j]/s;
            }
        }
        matprint(r,2,k);
        cout<<"r--------------------------------"<<endl;
        gettimeofday(&t2, 0);
        ctime[t*2+1] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
    }
    return;  
} 

__global__ void compute_n_pi(double *pi, double *n, double *r, int l, int k){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    for(int i = 0;i < l;++i){
        n[id] += r[i*k+id];
    }
    pi[id] = n[id]/(double)l;
}

__global__ void compute_pi(double *pi, double *n, int l, int k){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    pi[id] = n[id]/(double)l; 
}

__global__ void compute_div(double *m, double *n, int k, int d){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    for(int i = 0;i < d;i++){
        m[id*d+i] = m[id*d+i]/n[id];
    }
}

__global__ void compute_v(double *v, double *p, double *mu, double *r, int l, int k, int d){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k*l) return;
    int i = id/k, j = id%k;
    double *temp = create(d,1), *var = create(d,d), *trans = create(d,1);
    matsub(temp,&p[i*d],&mu[j*d],d,1);
    matmul(var,temp,transpose(trans,temp,d,1),d,1,d);
    for(int s = 0;s < d*d;++s){
        v[l*(j*d*d + s) + i] = var[s]*r[i*k+j];
    }
    free(temp); free(var); free(trans);
}

__host__ __device__ void helper_r(double *r, double *pi, double *p, double *mu, double *sigma, int l, int k, int d, int i){
    double s = 0.0;
    // for(int j = 0;j < k; ++j){
    //     if(i == 0 && j < 10) printf("%f ",r[i*k+j]);
    // }
    // if(i == 0) printf("\n");
    for(int j = 0;j < k; ++j){
        r[i*k+j] = pi[j]*norm(&p[i*d],&mu[j*d],&sigma[j*d*d],d);
        s+=r[i*k+j];
    }
    for(int j = 0;j < k;++j){
        r[i*k+j] = r[i*k+j]/s;
        // if(i == 0 && j < 10) printf("%f ",r[i*k+j]);
    }
    // if(i == 0) printf("\n");
}

__global__ void compute_r(double *r, double *pi, double *p, double *mu, double *sigma, int l, int k, int d){
    int i = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(i >= l) return;
    helper_r(r,pi,p,mu,sigma,l,k,d,i);
}

__global__ void gmatprint(double *a, int m, int n){
    matprint(a,m,n);
}

void cublas_atb(double *res, const double *a, const double *b, const int m, const int k, const int n, hipblasHandle_t &handle) {
    int lda=n,ldb=m,ldc=n;
    const double alf = 1.0;
    const double bet = 0.0;
    const double *alpha = &alf;
    const double *beta = &bet;
    hipblasStatus_t stat;
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, alpha, b, lda, a, ldb, beta, res, ldc);
    printf ("%d\n",stat);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#cublas multiply error\n");
        exit(1);
    }
}

void cublas_ab(double *res, const double *a, const double *b, const int m, const int k, const int n,hipblasHandle_t &handle) {
    int lda=n,ldb=k,ldc=n;
    const double alf = 1.0;
    const double bet = 0.0;
    const double *alpha = &alf;
    const double *beta = &bet;
    hipblasStatus_t stat;
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, b, lda, a, ldb, beta, res, ldc);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#cublas multiply error\n");
        exit(1);
    }
}

__global__ void gmatmul_ab(double *res, double *a, double *b, int m, int k, int n){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= m*n) return;
    int i = id/n, j = id%n;
    res[i*n+j] = 0.0;
    for(int l = 0;l < k;++l){
        res[i*n+j] += (a[i*k+l]*b[l*n+j]);
    }
}
__global__ void gmatmul_atb(double *res, double *a, double *b, int m, int k, int n){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= m*n) return;
    int i = id/n, j = id%n;
    res[i*n+j] = 0.0;
    for(int l = 0;l < k;++l){
        res[i*n+j] += (a[l*m+i]*b[l*n+j]);
    }
}
 
void gmix_gpu(double *cp,double *cr, int k, int iter, int l, int d, double *gtime){
    hipDeviceSetLimit(hipLimitStackSize, 1 << 25);

    // debugging
    // double *res, *a, *b;
    // hipMalloc(&res, 6*sizeof(double));
    // hipMalloc(&a, 6*sizeof(double));
    // hipMalloc(&b, 6*sizeof(double));
    // double ca[6] = {1.0,2.0,3.0,4.0,5.0,6.0}, cb[6] = {1.0,0.0,0.0,1.0,1.0,0.0};
    // hipMemcpy(a,ca,6*sizeof(double),hipMemcpyHostToDevice);
    // hipMemcpy(b,cb,6*sizeof(double),hipMemcpyHostToDevice);
    // cublas_atb(res, a, b, 3, 2, 3);
    // gmatprint<<<1,1>>>(res,3,3);
    

    double *n,*pi,*mu,*sigma;
    hipMalloc(&n,k*sizeof(double));
    hipMalloc(&pi,k*sizeof(double));
    hipMalloc(&mu,k*d*sizeof(double));
    hipMalloc(&sigma,k*d*d*sizeof(double));
    double *temp,*var,*trans,*lones;
    hipMalloc(&temp,d*sizeof(double));
    hipMalloc(&var,d*d*sizeof(double));
    hipMalloc(&trans,d*sizeof(double));
    hipMalloc(&lones,l*sizeof(double));
    double clo[l]; fill_n(clo, l, 1.0);
    hipMemcpy(lones,clo,l*sizeof(double),hipMemcpyHostToDevice);

    double *p,*r,*v;
    hipMalloc(&p,l*d*sizeof(double));
    hipMalloc(&r,l*k*sizeof(double));
    hipMalloc(&v,l*d*d*k*sizeof(double));
    hipMemcpy(p,cp,l*d*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(r,cr,l*k*sizeof(double),hipMemcpyHostToDevice);

    // hipblasHandle_t handle;
    // hipblasStatus_t stat = hipblasCreate(&handle);
    // if (stat != HIPBLAS_STATUS_SUCCESS) {
    //     printf ("#handle create error\n");
    //     exit(1);
    // }
    struct timeval t1, t2;
    int blockSize = 256;

    // rik -> P(aprior), pi -> alpha, 

    for(int t = 0; t < iter;++t){
        gettimeofday(&t1, 0);
        // M-step
        // cublas_atb(n,r,lones,k,l,1,handle);
        gmatmul_atb<<<(k/blockSize)+1,blockSize>>>(n,r,lones,k,l,1);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(n, 1, k);
        // hipDeviceSynchronize();
        // cout<<"n-----------------------------"<<endl;
        compute_pi<<<(k/blockSize)+1,blockSize>>>(pi,n,l,k);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(pi, 1, k);
        // hipDeviceSynchronize();
        // cout<<"pi-----------------------------"<<endl;
        // cublas_atb(mu, r, p, k, l, d,handle);
        gmatmul_atb<<<(k*d/blockSize)+1,blockSize>>>(mu,r,p,k,l,d);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(mu, 1, k*d);
        // hipDeviceSynchronize();
        // cout<<"mu-----------------------------"<<endl;
        compute_div<<<(k/blockSize)+1,blockSize>>>(mu,n,k,d);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(mu, 1, k*d);
        // hipDeviceSynchronize();
        // cout<<"mu-----------------------------"<<endl;
        compute_v<<<(k*l/blockSize)+1,blockSize>>>(v,p,mu,r,l,k,d);
        hipDeviceSynchronize();
        // cublas_ab(sigma, v, lones, d*d*k,l,1,handle);
        gmatmul_ab<<<(k*d*d/blockSize)+1,blockSize>>>(sigma,v,lones,d*d*k,l,1);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(sigma, k, d*d);
        // hipDeviceSynchronize();
        // cout<<"sigma-----------------------------"<<endl;
        compute_div<<<(k/blockSize)+1,blockSize>>>(sigma,n,k,d*d);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(sigma, k, d*d);
        // hipDeviceSynchronize();
        // cout<<"sigma-----------------------------"<<endl;
        gettimeofday(&t2, 0);
        gtime[t*2] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
        cout<<"GPU:"<<t<<endl;
        gettimeofday(&t1, 0);
        // E-step
        // gmatprint<<<1,1>>>(r, 1, k);
        // hipDeviceSynchronize();
        // cout<<"r-----------------------------"<<endl;
        // compute_r<<<l,1>>>(r, pi, p, mu, sigma, l, k, d);
        
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(r, 1, k);
        // hipDeviceSynchronize();
        // cout<<"r-----------------------------"<<endl;
        gettimeofday(&t2, 0);
        gtime[t*2+1] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
    }

    hipMemcpy(cr,r,l*k*sizeof(double),hipMemcpyDeviceToHost);

    // stat = hipblasDestroy(handle);
    // if (stat != HIPBLAS_STATUS_SUCCESS) {
    //     printf ("#handle destroy error\n");
    //     exit(1);
    // }
    hipFree(n); hipFree(pi); hipFree(mu); hipFree(sigma); 
    hipFree(p); hipFree(r); hipFree(v);
    hipFree(temp); hipFree(var); hipFree(trans); hipFree(lones);
    return;
}


