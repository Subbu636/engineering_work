#include "hip/hip_runtime.h"
#include "util.h"

__host__ __device__ double *create(int m, int n){
    return (double*)malloc(sizeof(double)*n*m);
}

__host__ __device__ void init(double *v, int m, int n, double val){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            v[i*n+j] = val;
        }
    }
}

__host__ __device__ void copy(double *a, double *b, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            a[i*m+j] = b[i*m+j];
        }
    }
}


__host__ __device__ double *transpose(double *trans, double *v, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            trans[j*m+i] = v[i*n+j];
        }
    }
    return trans;
}

__host__ __device__ double *matsub(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]-v2[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matmul(double *res, double *a, double *b, int m, int s, int n){
    init(res, m, n, 0.0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < s; k++) res[i*n+j] += a[i*s+k]*b[k*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matadd(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]+v2[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matsmul(double *res, double *v, double val, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = val*v[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *cofactor(double *res, double *a, int p, int q, int n)
{
    int r = 0, c = 0;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i != p && j != q)
            {
                res[r*(n-1)+(c++)] = a[i*n+j];
                if (c == n - 1)
                {
                    c = 0;
                    r++;
                }
            }
        }
    }
    return res;
}

__host__ __device__ double matdet(double *a, int n)
{
    double res = 0.0;
    if (n == 1) return a[0];
    double s = 1.0, *temp = create(n-1,n-1);
    for (int i = 0; i < n; i++)
    {
        res += (s*a[i]*matdet(cofactor(temp,a,0,i,n),n-1));
        s = -1.0*s;
    }
    // free(temp);
    return res;
}

__host__ __device__ double *matadj(double *res, double *a, int n)
{
    if(n == 1){
        res[0] = 1.0;
        return res;
    }
    double s = 1.0, *temp = create(n,n);
    for(int i = 0;i < n;i++)
    {
        for(int j = 0;j < n;++j)
        {
            if((i+j)%2 == 0) s = 1.0;
            else s = -1.0;
            res[j*n+i] = (s)*(matdet(cofactor(temp,a,i,j,n), n-1));
        }
    }
    free(temp);
    return res;
}

__host__ __device__ double *matinv(double *res, double *a, int n)
{
    double d = matdet(a,n);
    if (d == 0.0)
    {
        printf("#singular - inverse not possible\n");
        assert(false);
    }
    double *adj = create(n,n);
    matadj(adj,a,n);
    for(int i = 0;i < n;++i){
        for(int j = 0;j < n;++j){
            res[i*n+j] = adj[i*n+j]/d;
        }
    }
    free(adj);
    return res;
}

__host__ __device__ double norm(double *x, double *mu, double *sigma, int d){
    double deno = sqrt(pow((double)(2.0*3.14159),(double)d)*matdet(sigma,d));
    double *dif = create(d,1),*temp = create(d,1),*inv = create(d,d),val[1],*trans = create(d,1);
    matsub(dif,x,mu,d,1);
    double expo = (-0.5)*(matmul(val,matmul(temp,transpose(trans,dif,d,1),matinv(inv,sigma,d),1,d,d),dif,1,d,1)[0]);
    // printf("%lf %lf\n",matdet(sigma,d),inv[0]);
    free(dif); free(temp); free(inv); free(trans);
    return exp(expo)/deno;
}

__host__ __device__ void matprint(double *a, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            printf("%lf ",a[i*n+j]);
        }
        printf ("\n");
    }
}


void gmix_cpu(double *p,double *r, int k, int iter, int l, int d, double *ctime){

    // Debugging
    // double arr[4] = {2.0,1.0,1.0,2.0};
    // double mu[2] = {1.0, 1.0}, x[2] = {1.0, 0.0};
    // cout<<norm(x,mu,arr,2)<<endl;

    double n[k],pi[k],mu[k*d],sigma[k*d*d];
    double temp[d], var[d*d],trans[d];
    struct timeval t1, t2;

    for(int t = 0;t < iter;++t){
        gettimeofday(&t1, 0);
        // M-step
        init(n,1,k,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                n[i] += r[j*k+i];
            }
        }
        // matprint(n,1,k);
        // cout<<"n--------------------------------"<<endl;
        for(int i = 0;i < k;++i){
            pi[i] = n[i]/(double)l;
        }
        // matprint(pi,1,k);
        // cout<<"pi--------------------------------"<<endl;
        init(mu,k,d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matadd(&mu[i*d],&mu[i*d],matsmul(temp,&p[j*d],r[j*k+i],d,1),d,1);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&mu[i*d],&mu[i*d],1/n[i],d,1);
        }
        // matprint(mu,1,k*d);
        // cout<<"mu--------------------------------"<<endl;
        init(sigma,k,d*d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matsub(temp,&p[j*d],&mu[i*d],d,1);
                matmul(var,temp,transpose(trans,temp,d,1),d,1,d);
                matsmul(var,var,r[j*k+i],d,d);
                matadd(&sigma[i*d*d],&sigma[i*d*d],var,d,d);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&sigma[i*d*d],&sigma[i*d*d],1/n[i],d,d);
        }
        // matprint(sigma,1,k*d*d);
        // cout<<"sigma--------------------------------"<<endl;
        gettimeofday(&t2, 0);
        ctime[t*2] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
        gettimeofday(&t1, 0);
        // E-step
        for(int i = 0;i < l; ++i){
            double s = 0;
            for(int j = 0;j < k; ++j){
                r[i*k+j] = pi[j]*norm(&p[i*d],&mu[j*d],&sigma[j*d*d],d);
                s+=r[i*k+j];
            }
            for(int j = 0;j < k;++j){
                r[i*k+j] = r[i*k+j]/s;
            }
        }
        // matprint(r,1,k);
        // cout<<"r--------------------------------"<<endl;
        gettimeofday(&t2, 0);
        ctime[t*2+1] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
    }
    return;  
} 

__global__ void compute_n_pi(double *pi, double *n, double *r, int l, int k){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    for(int i = 0;i < l;++i){
        n[id] += r[i*k+id];
    }
    pi[id] = n[id]/(double)l;
}

__global__ void compute_pi(double *pi, double *n, int l, int k){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    pi[id] = n[id]/(double)l; 
}

__global__ void compute_div(double *m, double *n, int k, int d){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    for(int i = 0;i < d;i++){
        m[id*d+i] = m[id*d+i]/n[id];
    }
}

__global__ void compute_v(double *v, double *p, double *mu, double *r, int l, int k, int d){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k*l) return;
    int i = id/k, j = id%k;
    double *temp = create(d,1), *var = create(d,d), *trans = create(d,1);
    matsub(temp,&p[i*d],&mu[j*d],d,1);
    matmul(var,temp,transpose(trans,temp,d,1),d,1,d);
    for(int s = 0;s < d*d;++s){
        v[l*(j*d*d + s) + i] = var[s]*r[i*k+j];
    }
    free(temp); free(var); free(trans);
}

__global__ void compute_r(double *r, double *pi, double *p, double *mu, double *sigma, int l, int k, int d){
    int i = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(i >= l) return;
    double s = 0.0;
    for(int j = 0;j < k; ++j){
        r[i*k+j] = pi[j]*norm(&p[i*d],&mu[j*d],&sigma[j*d*d],d);
        s+=r[i*k+j];
    }
    for(int j = 0;j < k;++j){
        r[i*k+j] = r[i*k+j]/s;
    }
}

__global__ void gmatprint(double *a, int m, int n){
    matprint(a,m,n);
}

void cublas_atb(double *res, const double *a, const double *b, const int m, const int k, const int n, hipblasHandle_t &handle) {
    int lda=n,ldb=m,ldc=n;
    const double alf = 1.0;
    const double bet = 0.0;
    const double *alpha = &alf;
    const double *beta = &bet;
    hipblasStatus_t stat;
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, alpha, b, lda, a, ldb, beta, res, ldc);
    printf ("%d\n",stat);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#cublas multiply error\n");
        exit(1);
    }
}

void cublas_ab(double *res, const double *a, const double *b, const int m, const int k, const int n,hipblasHandle_t &handle) {
    int lda=n,ldb=k,ldc=n;
    const double alf = 1.0;
    const double bet = 0.0;
    const double *alpha = &alf;
    const double *beta = &bet;
    hipblasStatus_t stat;
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, b, lda, a, ldb, beta, res, ldc);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#cublas multiply error\n");
        exit(1);
    }
}
 
void gmix_gpu(double *cp,double *cr, int k, int iter, int l, int d, double *gtime){
    hipDeviceSetLimit(hipLimitStackSize, 1 << 25);
    // debugging
    // double *res, *a, *b;
    // hipMalloc(&res, 6*sizeof(double));
    // hipMalloc(&a, 6*sizeof(double));
    // hipMalloc(&b, 6*sizeof(double));
    // double ca[6] = {1.0,2.0,3.0,4.0,5.0,6.0}, cb[6] = {1.0,0.0,0.0,1.0,1.0,0.0};
    // hipMemcpy(a,ca,6*sizeof(double),hipMemcpyHostToDevice);
    // hipMemcpy(b,cb,6*sizeof(double),hipMemcpyHostToDevice);
    // cublas_atb(res, a, b, 3, 2, 3);
    // gmatprint<<<1,1>>>(res,3,3);
    

    double *n,*pi,*mu,*sigma;
    hipMalloc(&n,k*sizeof(double));
    hipMalloc(&pi,k*sizeof(double));
    hipMalloc(&mu,k*d*sizeof(double));
    hipMalloc(&sigma,k*d*d*sizeof(double));
    double *temp,*var,*trans,*lones;
    hipMalloc(&temp,d*sizeof(double));
    hipMalloc(&var,d*d*sizeof(double));
    hipMalloc(&trans,d*sizeof(double));
    hipMalloc(&lones,l*sizeof(double));
    double clo[l]; fill_n(clo, l, 1.0);
    hipMemcpy(lones,clo,l*sizeof(double),hipMemcpyHostToDevice);

    double *p,*r,*v;
    hipMalloc(&p,l*d*sizeof(double));
    hipMalloc(&r,l*k*sizeof(double));
    hipMalloc(&v,l*d*d*k*sizeof(double));
    hipMemcpy(p,cp,l*d*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(r,cr,l*k*sizeof(double),hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#handle create error\n");
        exit(1);
    }
    struct timeval t1, t2;

    // rik -> P(aprior), pi -> alpha, 

    for(int t = 0; t < iter;++t){
        gettimeofday(&t1, 0);
        // M-step
        cublas_atb(n,r,lones,k,l,1,handle);
        // gmatprint<<<1,1>>>(n, 1, k);
        // hipDeviceSynchronize();
        // cout<<"n-----------------------------"<<endl;
        compute_pi<<<k,1>>>(pi,n,l,k);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(pi, 1, k);
        // hipDeviceSynchronize();
        // cout<<"pi-----------------------------"<<endl;
        cublas_atb(mu, r, p, k, l, d,handle);
        // gmatprint<<<1,1>>>(mu, 1, k*d);
        // hipDeviceSynchronize();
        // cout<<"mu-----------------------------"<<endl;
        compute_div<<<k,1>>>(mu,n,k,d);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(mu, 1, k*d);
        // hipDeviceSynchronize();
        // cout<<"mu-----------------------------"<<endl;
        compute_v<<<l*k,1>>>(v,p,mu,r,l,k,d);
        hipDeviceSynchronize();
        cublas_ab(sigma, v, lones, d*d*k,l,1,handle);
        // gmatprint<<<1,1>>>(sigma, k, d*d);
        // hipDeviceSynchronize();
        // cout<<"sigma-----------------------------"<<endl;
        compute_div<<<k,1>>>(sigma,n,k,d*d);
        hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(sigma, k, d*d);
        // hipDeviceSynchronize();
        // cout<<"sigma-----------------------------"<<endl;
        gettimeofday(&t2, 0);
        gtime[t*2] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
        gettimeofday(&t1, 0);
        // E-step
        compute_r<<<l,1>>>(r, pi, p, mu, sigma, l, k, d);
        // gmatprint<<<1,1>>>(r, 1, k);
        // hipDeviceSynchronize();
        // cout<<"r0-----------------------------"<<endl;
        gettimeofday(&t2, 0);
        gtime[t*2+1] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);

    }
    stat = hipblasDestroy(handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#handle destroy error\n");
        exit(1);
    }
    hipMemcpy(cr,r,l*k*sizeof(double),hipMemcpyDeviceToHost);
    return;
}


