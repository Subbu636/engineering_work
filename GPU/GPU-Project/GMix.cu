#include "hip/hip_runtime.h"
#include "util.h"


double *create(int m, int n){
    return (double*)malloc(sizeof(double)*n*m);
}

void init(double *v, int m, int n, double val){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            v[i*n+j] = val;
        }
    }
}

void copy(double *a, double *b, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            a[i*m+j] = b[i*m+j];
        }
    }
}


double *transpose(double *trans, double *v, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            trans[j*m+i] = v[i*n+j];
        }
    }
    return trans;
}

double *matsub(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]-v2[i*n+j];
        }
    }
    return res;
}

double *matmul(double *res, double *a, double *b, int m, int s, int n){
    init(res, m, n, 0.0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < s; k++) res[i*n+j] += a[i*s+k]*b[k*n+j];
        }
    }
    return res;
}

double *matadd(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]+v2[i*n+j];
        }
    }
    return res;
}

double *matsmul(double *res, double *v, double val, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = val*v[i*n+j];
        }
    }
    return res;
}

double *cofactor(double *res, double *a, int p, int q, int n)
{
    int r = 0, c = 0;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i != p && j != q)
            {
                res[r*(n-1)+(c++)] = a[i*n+j];
                if (c == n - 1)
                {
                    c = 0;
                    r++;
                }
            }
        }
    }
    return res;
}

double matdet(double *a, int n)
{
    double res = 0.0;
    if (n == 1) return a[0];
    double s = 1.0,temp[(n-1)*(n-1)];
    for (int i = 0; i < n; i++)
    {
        res += (s*a[i]*matdet(cofactor(temp,a,0,i,n),n-1));
        s = -1.0*s;
    }
    return res;
}

double *matadj(double *res, double *a, int n)
{
    if(n == 1){
        res[0] = 1.0;
        return res;
    }
    double s = 1.0, temp[n*n];
    for(int i = 0;i < n;i++)
    {
        for(int j = 0;j < n;++j)
        {
            if((i+j)%2 == 0) s = 1.0;
            else s = -1.0;
            res[j*n+i] = (s)*(matdet(cofactor(temp,a,i,j,n), n-1));
        }
    }
    return res;
}

double *matinv(double *res, double *a, int n)
{
    double d = matdet(a,n);
    if (d == 0.0)
    {
        cout<<"#singular - inverse not possible"<<endl;
        assert(false);
    }
    double adj[n*n];
    matadj(adj,a,n);
    for(int i = 0;i < n;++i){
        for(int j = 0;j < n;++j){
            res[i*n+j] = adj[i*n+j]/d;
        }
    }
    return res;
}

double norm(double *x, double *mu, double *sigma, int d){
    double deno = sqrt(pow((double)(2.0*M_PI),(double)d)*matdet(sigma,d));
    double dif[d],temp[d],inv[d*d],val[1],trans[d];
    matsub(dif,x,mu,d,1);
    double expo = (-0.5)*(matmul(val,matmul(temp,transpose(trans,dif,d,1),matinv(inv,sigma,d),1,d,d),dif,1,d,1)[0]);
    // cout<<expo<<endl;
    return exp(expo)/deno;
}

void matprint(double *a, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            cout<<a[i*n+j]<<" ";
        }
        cout<<endl;
    }
}


void gmix_cpu(double *p,double *r, int k, int iter, int l, int d){

    // Debugging
    // double arr[4] = {2.0,1.0,1.0,2.0};
    // double mu[2] = {1.0, 1.0}, x[2] = {1.0, 0.0};
    // cout<<norm(x,mu,arr,2)<<endl;

    double n[k],pi[k],mu[k*d],sigma[k*d*d];
    double temp[d], var[d*d],trans[d];

    for(int t = 0;t < iter;++t){
        // M-step
        init(n,1,k,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                n[i] += r[j*k+i];
            }
        }
        for(int i = 0;i < k;++i){
            pi[i] = n[i]/(double)l;
        }
        init(mu,k,d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matadd(&mu[i*d],&mu[i*d],matsmul(temp,&p[j*d],r[j*k+i],d,1),d,1);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&mu[i*d],&mu[i*d],1/n[i],d,1);
        }
        init(sigma,k,d*d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matsub(temp,&p[j*d],&mu[i*d],d,1);
                matmul(var,temp,transpose(trans,temp,d,1),d,1,d);
                matsmul(var,var,r[j*k+i],d,d);
                matadd(&sigma[i*d*d],&sigma[i*d*d],var,d,d);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&sigma[i*d*d],&sigma[i*d*d],1/n[i],d,d);
        }
        // E-step
        for(int i = 0;i < l; ++i){
            double s = 0;
            for(int j = 0;j < k; ++j){
                r[i*k+j] = pi[j]*norm(&p[i*d],&mu[j*d],&sigma[j*d*d],d);
                s+=r[i*k+j];
            }
            for(int j = 0;j < k;++j){
                r[i*k+j] = r[i*k+j]/s;
            }
        }
    }
    return;  
} 

__global__ void compute_n_pi(double *pi, double *n, double *r, int l, int k){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    for(int i = 0;i < l;++i){
        n[id] += r[i*k+id];
    }
    pi[id] = n[id]/(double)l;
}

void gmatmul(double *res, double *a, double *b, int m, int s, int n){
    double x = 1.0, y = 0.0;
    hipComplex *gx,*gy;
    hipMalloc(&gx,sizeof(double));
    hipMalloc(&gy,sizeof(double));
    hipMemcpy(gx,&x,sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(gy,&y,sizeof(double),hipMemcpyHostToDevice);
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("create problem\n");
    }
    stat = cublasCgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, s, gx, (hipComplex*)a, m, (hipComplex*)b, s, gy,(hipComplex*)res, m);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("multiply problem\n");
    }
    return;
}
 
void gmix_gpu(double *p,double *r, int k, int iter, int l, int d){

    // debugging
    double *res, *a, *b;
    hipMalloc(&res, d*d*sizeof(double));
    hipMalloc(&a, d*d*sizeof(double));
    hipMalloc(&b, d*d*sizeof(double));
    double ca[4] = {1.0,2.0,3.0,4.0}, cb[4] = {1.0,0.0,0.0,1.0};
    hipMemcpy(a,ca,d*d*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(b,cb,d*d*sizeof(double),hipMemcpyHostToDevice);
    gmatmul(res,a,b,2,2,2);
    double cres[4];
    hipMemcpy(cres,res,d*d*sizeof(double),hipMemcpyDeviceToHost);
    matprint(cres,2,2);

    // double *n,*pi,*mu,*sigma;
    // hipMalloc(&n,k*sizeof(double));
    // hipMalloc(&pi,k*sizeof(double));
    // hipMalloc(&mu,k*d*sizeof(double));
    // hipMalloc(&sigma,k*d*d*sizeof(double));
    // double *temp,*var,*trans;
    // hipMalloc(&temp,d*sizeof(double));
    // hipMalloc(&var,d*d*sizeof(double));
    // hipMalloc(&trans,d*sizeof(double));

    // double *gp,*gr;
    // hipMalloc(&gp,l*d*sizeof(double));
    // hipMalloc(&gr,l*k*sizeof(double));
    // hipMemcpy(gp,p,l*d*sizeof(double),hipMemcpyHostToDevice);
    // hipMemcpy(gr,r,l*k*sizeof(double),hipMemcpyHostToDevice);

    // // rik -> aprior or P, pi -> alpha, 

    // for(int t = 0; t < iter;++t){
    //     compute_n_pi<<<k,1>>>(pi,n,r,k,l);
    //     hipDeviceSynchronize();

    // }
    return;
}


