#include "hip/hip_runtime.h"
#include "util.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__ __device__ double *create(int m, int n){
    return (double*)malloc(sizeof(double)*n*m);
}

__host__ __device__ void init(double *v, int m, int n, double val){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            v[i*n+j] = val;
        }
    }
}

__host__ __device__ void copy(double *a, double *b, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            a[i*m+j] = b[i*m+j];
        }
    }
}


__host__ __device__ double *transpose(double *trans, double *v, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            trans[j*m+i] = v[i*n+j];
        }
    }
    return trans;
}

__host__ __device__ double *matsub(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]-v2[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matmul(double *res, double *a, double *b, int m, int s, int n){
    init(res, m, n, 0.0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < s; k++) res[i*n+j] += a[i*s+k]*b[k*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matadd(double *res, double *v1, double *v2, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = v1[i*n+j]+v2[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double *matsmul(double *res, double *v, double val, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;++j){
            res[i*n+j] = val*v[i*n+j];
        }
    }
    return res;
}

__host__ __device__ double norm(double *x, double *mu, double *sigma, int d){
    double deno = sqrt((double)(2.0*3.14159));
    double expo = 1.0;
    for(int i = 0;i < d;++i){
        expo *= (exp(-0.5*(x[i]-mu[i])*(x[i]-mu[i])/sigma[i*d+i])/(deno*sqrt(sigma[i*d+i])));
    }
    return expo;
}

__host__ __device__ void matprint(double *a, int m, int n){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            printf("%f ",a[i*n+j]);
        }
        printf ("\n");
    }
}

void gmix_cpu(double *p,double *r, int k, int iter, int l, int d, double *ctime){

    // Debugging
    // double arr[4] = {2.0,1.0,1.0,2.0};
    // double mu[2] = {1.0, 1.0}, x[2] = {1.0, 0.0};
    // cout<<norm(x,mu,arr,2)<<endl;

    double n[k],pi[k],mu[k*d],sigma[k*d*d];
    double temp[d], var[d*d],trans[d];
    struct timeval t1, t2;

    // matprint(r,2,k);
    // cout<<"r----------------------------"<<endl;

    for(int t = 0;t < iter;++t){
        gettimeofday(&t1, 0);
        // M-step
        init(n,1,k,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                n[i] += r[j*k+i];
            }
        }
        // matprint(n,1,k);
        // cout<<"n--------------------------------"<<endl;
        for(int i = 0;i < k;++i){
            pi[i] = n[i]/(double)l;
        }
        // matprint(pi,1,k);
        // cout<<"pi--------------------------------"<<endl;
        init(mu,k,d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matadd(&mu[i*d],&mu[i*d],matsmul(temp,&p[j*d],r[j*k+i],d,1),d,1);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&mu[i*d],&mu[i*d],1/n[i],d,1);
        }
        // matprint(mu,1,k*d);
        // cout<<"mu--------------------------------"<<endl;
        init(sigma,k,d*d,0.0);
        for(int i = 0;i < k;++i){
            for(int j = 0;j < l;++j){
                matsub(temp,&p[j*d],&mu[i*d],d,1);
                matmul(var,temp,transpose(trans,temp,d,1),d,1,d);
                matsmul(var,var,r[j*k+i],d,d);
                matadd(&sigma[i*d*d],&sigma[i*d*d],var,d,d);
            }
        }
        for(int i = 0;i < k;++i){
            matsmul(&sigma[i*d*d],&sigma[i*d*d],1/n[i],d,d);
        }
        // matprint(sigma,1,k*d*d);
        // cout<<"sigma--------------------------------"<<endl;
        gettimeofday(&t2, 0);
        ctime[t*2] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
        cout<<"CPU:"<<t<<endl;
        gettimeofday(&t1, 0);
        // E-step
        for(int i = 0;i < l; ++i){
            double s = 0;
            for(int j = 0;j < k; ++j){
                r[i*k+j] = pi[j]*norm(&p[i*d],&mu[j*d],&sigma[j*d*d],d);
                s+=r[i*k+j];
            }
            assert(s != 0.0);
            for(int j = 0;j < k;++j){
                r[i*k+j] = r[i*k+j]/s;
            }
        }
        // matprint(r,1,k);
        // cout<<"r--------------------------------"<<endl;
        gettimeofday(&t2, 0);
        ctime[t*2+1] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
    }
    return;  
} 



__global__ void compute_n_pi(double *pi, double *n, double *r, int l, int k){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    n[id] = 0.0;
    for(int i = 0;i < l;++i){
        n[id] += r[i*k+id];
    }
    pi[id] = n[id]/(double)l;
}

__global__ void compute_pi(double *pi, double *n, int l, int k){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    pi[id] = n[id]/(double)l; 
}

__global__ void compute_div(double *m, double *n, int k, int d){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k) return;
    for(int i = 0;i < d;i++){
        m[id*d+i] = m[id*d+i]/n[id];
    }
}

__global__ void compute_v(double *v, double *p, double *mu, double *r, int l, int k, int d){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= k*l*d*d) return;
    // int y = id%d, x = (id/d)%d, j = (id/(d*d))%k, i = (id/(d*d*k))%l;
    int y = (id/(l*k*d))%d, x = (id/(l*k))%d, j = (id/l)%k, i = id%l;
    // for (int x = 0; x < d; x++) {
    //     for (int y = 0; y < d; y++) {
    //         int s = x*d+y;
    //         v[l*(j*d*d + s) + i] = (p[i*d+x]-mu[j*d+x])*(p[i*d+y]-mu[j*d+y])*r[i*k+j];
    //     }
    // }
    v[l*(j*d*d + x*d + y) + i] = (p[i*d+x]-mu[j*d+x])*(p[i*d+y]-mu[j*d+y])*r[i*k+j];
}

__global__ void compute_r(double *r, double *pi, double *p, double *mu, double *sigma, int l, int k, int d){
    int i = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(i >= l) return;
    double s = 0.0;
    for(int j = 0;j < k; ++j){
        r[i*k+j] = pi[j]*norm(&p[i*d],&mu[j*d],&sigma[j*d*d],d);
        s+=r[i*k+j];
    }
    for(int j = 0;j < k;++j){
        r[i*k+j] = r[i*k+j]/s;
    }
}

__global__ void gmatprint(double *a, int m, int n){
    matprint(a,m,n);
}

void cublas_atb(double *res, const double *a, const double *b, const int m, const int k, const int n, hipblasHandle_t &handle) {
    int lda=n,ldb=m,ldc=n;
    const double alf = 1.0;
    const double bet = 0.0;
    const double *alpha = &alf;
    const double *beta = &bet;
    hipblasStatus_t stat;
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, alpha, b, lda, a, ldb, beta, res, ldc);
    printf ("%d\n",stat);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#cublas multiply error\n");
        exit(1);
    }
}

void cublas_ab(double *res, const double *a, const double *b, const int m, const int k, const int n,hipblasHandle_t &handle) {
    int lda=n,ldb=k,ldc=n;
    const double alf = 1.0;
    const double bet = 0.0;
    const double *alpha = &alf;
    const double *beta = &bet;
    hipblasStatus_t stat;
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, b, lda, a, ldb, beta, res, ldc);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("#cublas multiply error\n");
        exit(1);
    }
}

__global__ void gmatmul_ab(double *res, double *a, double *b, int m, int k, int n){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= m*n) return;
    int i = id/n, j = id%n;
    res[i*n+j] = 0.0;
    for(int l = 0;l < k;++l){
        res[i*n+j] += (a[i*k+l]*b[l*n+j]);
    }
}
__global__ void gmatmul_atb(double *res, double *a, double *b, int m, int k, int n){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= m*n) return;
    int i = id/n, j = id%n;
    res[i*n+j] = 0.0;
    for(int l = 0;l < k;++l){
        res[i*n+j] += (a[l*m+i]*b[l*n+j]);
    }
}

__global__ void gmatmul_atb_mu(double *res, double *a, double *b, double *nn, int m, int k, int n){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= m*n) return;
    int i = id/n, j = id%n;
    res[i*n+j] = 0.0;
    for(int l = 0;l < k;++l){
        res[i*n+j] += (a[l*m+i]*b[l*n+j]);
    }
    res[i*n+j] = res[i*n+j]/nn[i];
}

__global__ void gmatmul_ab_sigma(double *res, double *a, double *b, double *nn, int m, int k, int n,int d){
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= m*n) return;
    int i = id/n, j = id%n;
    res[i*n+j] = 0.0;
    for(int l = 0;l < k;++l){
        res[i*n+j] += (a[i*k+l]*b[l*n+j]);
    }
    res[i*n+j] = res[i*n+j]/nn[i/(d*d)];
}
 
void gmix_gpu(double *cp,double *cr, int k, int iter, int l, int d, double *gtime){
    hipDeviceSetLimit(hipLimitStackSize, 1 << 16);
    gpuErrchk(hipPeekAtLastError());
    // debugging
    // double *res, *a, *b;
    // hipMalloc(&res, 6*sizeof(double));
    // hipMalloc(&a, 6*sizeof(double));
    // hipMalloc(&b, 6*sizeof(double));
    // double ca[6] = {1.0,2.0,3.0,4.0,5.0,6.0}, cb[6] = {1.0,0.0,0.0,1.0,1.0,0.0};
    // hipMemcpy(a,ca,6*sizeof(double),hipMemcpyHostToDevice);
    // hipMemcpy(b,cb,6*sizeof(double),hipMemcpyHostToDevice);
    // cublas_atb(res, a, b, 3, 2, 3);
    // gmatprint<<<1,1>>>(res,3,3);
    

    double *n,*pi,*mu,*sigma;
    hipMalloc(&n,k*sizeof(double));
    hipMalloc(&pi,k*sizeof(double));
    hipMalloc(&mu,k*d*sizeof(double));
    hipMalloc(&sigma,k*d*d*sizeof(double));
    double *temp,*var,*trans,*lones;
    hipMalloc(&temp,d*sizeof(double));
    hipMalloc(&var,d*d*sizeof(double));
    hipMalloc(&trans,d*sizeof(double));
    hipMalloc(&lones,l*sizeof(double));
    double clo[l]; fill_n(clo, l, 1.0);
    hipMemcpy(lones,clo,l*sizeof(double),hipMemcpyHostToDevice);

    double *p,*r,*v;
    hipMalloc(&p,l*d*sizeof(double));
    hipMalloc(&r,l*k*sizeof(double));
    hipMalloc(&v,l*d*d*k*sizeof(double));
    hipMemcpy(p,cp,l*d*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(r,cr,l*k*sizeof(double),hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());

    // hipblasHandle_t handle;
    // hipblasStatus_t stat = hipblasCreate(&handle);
    // if (stat != HIPBLAS_STATUS_SUCCESS) {
    //     printf ("#handle create error\n");
    //     exit(1);
    // }
    struct timeval t1, t2;
    int blockSize = 256;

    // rik -> P(aprior), pi -> alpha, 

    for(int t = 0; t < iter;++t){
        gettimeofday(&t1, 0);
        // M-step
        // cublas_atb(n,r,lones,k,l,1,handle);
        compute_n_pi<<<(k),1>>>(pi, n, r, l, k);
        // gmatmul_atb<<<(k/blockSize)+1,blockSize>>>(n,r,lones,k,l,1);
        hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
        // gmatprint<<<1,1>>>(n, 1, k);
        // hipDeviceSynchronize();
        // cout<<"n-----------------------------"<<endl;
        // compute_pi<<<(k/blockSize)+1,blockSize>>>(pi,n,l,k);
        // hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(pi, 1, k);
        // hipDeviceSynchronize();
        // cout<<"pi-----------------------------"<<endl;
        // cublas_atb(mu, r, p, k, l, d,handle);
        gmatmul_atb_mu<<<(k*d),1>>>(mu,r,p,n,k,l,d);
        // gmatmul_atb<<<(k*d/blockSize)+1,blockSize>>>(mu,r,p,k,l,d);
        hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
        // gmatprint<<<1,1>>>(mu, 1, k*d);
        // hipDeviceSynchronize();
        // cout<<"mu-----------------------------"<<endl;
        // compute_div<<<(k/blockSize)+1,blockSize>>>(mu,n,k,d);
        // hipDeviceSynchronize();
        // gmatprint<<<1,1>>>(mu, 1, k*d);
        // hipDeviceSynchronize();
        // cout<<"mu-----------------------------"<<endl;
        compute_v<<<(k*l*d*d/blockSize)+1,blockSize>>>(v,p,mu,r,l,k,d);
        hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
        // gmatprint<<<1,1>>>(v, 1, k);
        // hipDeviceSynchronize();
        // cout<<"v-----------------------------"<<endl;
        // cublas_ab(sigma, v, lones, d*d*k,l,1,handle);
        gmatmul_ab_sigma<<<(k*d*d),1>>>(sigma,v,lones,n,d*d*k,l,1,d);
        // gmatmul_ab<<<(k*d*d/blockSize)+1,blockSize>>>(sigma,v,lones,d*d*k,l,1);
        hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
        // gmatprint<<<1,1>>>(sigma, k, d*d);
        // hipDeviceSynchronize();
        // cout<<"sigma-----------------------------"<<endl;
        // compute_div<<<(k/blockSize)+1,blockSize>>>(sigma,n,k,d*d);
        // hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
        // gmatprint<<<1,1>>>(sigma, k, d*d);
        // hipDeviceSynchronize();
        // cout<<"sigma-----------------------------"<<endl;
        gettimeofday(&t2, 0);
        gtime[t*2] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
        gpuErrchk(hipPeekAtLastError());
        cout<<"GPU:"<<t<<endl;
        gettimeofday(&t1, 0);
        // E-step
        // gmatprint<<<1,1>>>(r, 1, k);
        // hipDeviceSynchronize();
        // cout<<"r-----------------------------"<<endl;
        compute_r<<<(l/blockSize)+1,blockSize>>>(r, pi, p, mu, sigma, l, k, d);
        hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
        // gmatprint<<<1,1>>>(r, 1, 10);
        // hipDeviceSynchronize();
        // cout<<"r-----------------------------"<<endl;
        gettimeofday(&t2, 0);
        gtime[t*2+1] = ((double)(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0);
        gpuErrchk(hipPeekAtLastError());
    }

    hipMemcpy(cr,r,l*k*sizeof(double),hipMemcpyDeviceToHost);

    // stat = hipblasDestroy(handle);
    // if (stat != HIPBLAS_STATUS_SUCCESS) {
    //     printf ("#handle destroy error\n");
    //     exit(1);
    // }
    hipFree(n); hipFree(pi); hipFree(mu); hipFree(sigma); 
    hipFree(p); hipFree(r); hipFree(v);
    hipFree(temp); hipFree(var); hipFree(trans); hipFree(lones);
    gpuErrchk(hipPeekAtLastError());
    return;
}


