#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <sys/time.h> 
#include "util.h"
using namespace std;

int main(int argc,char **argv){
    int k = ;
    struct timeval t1, t2;
    char *fname = argv[1];
    vector <Point> points;
    read_file(fname,points);
    int num_points = points.size();
    Points** gpupoints; 
    Points* gpumeans;
    int* gpulabels;
    //Updating above variables
    int iter = 1000;
    // cpu
    gettimeofday(&t1, 0);
    kmeans_cpu(points,iter);
    gettimeofday(&t2, 0);
    double ct = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("CPU Time taken: %.6f ms\n", ct);
    // gpu
    gettimeofday(&t1, 0);
    kmeans_gpu(gpupoints,gpumeans,gpulabels,iter,num_points,k);
    gettimeofday(&t2, 0);
    double gt = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("GPU Time taken: %.6f ms\n", gt);
}


