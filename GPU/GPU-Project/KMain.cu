#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <sys/time.h> 
#include "util.h"
using namespace std;
Point* read(char *filename, int* num){
    FILE *file;
    file = fopen(filename, "r");
    if (!file)  {
        printf("#cannot open input file\n");
        return NULL;
    }
    vector<Point> points;
    float x,y;
    int n = 0;
    while(fscanf(file,"%f %f",&x,&y) != EOF){
        Point p;
        p.x = x;
        p.y = y;
        points.push_back(p);
        n++;
    }
     Point* ps = (Point*) malloc(n*sizeof(Point));
     for(int i=0;i<n;i++)
     {
         ps[i] = points[i];
     }
     num[0] = n;
     return ps;
}

int main(int argc,char **argv){
    int seed = 1234;
    srand(1234);
    struct timeval t1, t2;
    char *fname = argv[1];

    if(fname == NULL)
    {
        printf("give the input file\n");
        return 0;
    }
    char *oname = argv[2];
    
    if (oname == NULL)
    {
        printf("give the output file\n");
        return 0;
    }
    
    FILE *outfile;
    outfile = fopen(oname, "w");
    char* kchar = argv[3];
    if(kchar == NULL) 
    {
        printf("Input K\n");
        return 0;
    } 
    int k = atoi(kchar);

    Point* points;
    int p[1];
    p[0] = 0;
    points = read(fname,p);
    
    // for(int i=0;i<100;i++)
    // {
    //     printf("%f %f\n", points[i].x,points[i].y);
    // }

    int num_points = p[0];
    printf("Points Allocated\n");

    vector<int> ind;
    for(int i=0;i<k;i++) ind.push_back(i*(num_points/k));
    // for(int i=0;i<num_points;i++) random_shuffle(0, n);   random shuffle taking a lot time for 10^5 so used above method 
    Point* gpupoints;
    hipMalloc(&gpupoints, num_points*sizeof(Point));
    printf("GPU Points Allocated\n");
    
    hipMemcpy(gpupoints,points,num_points*sizeof(Point),hipMemcpyHostToDevice); 
    printf("GPU Points Copied\n");

    Point* means = (Point*) malloc(k*sizeof(Point));
    int* labels = (int*) malloc(num_points*sizeof(int));
    printf("Means and Labels Allocated\n");

    
    // Naive Sharding Initialization
    for(int i=0;i<num_points;i++)
    {
        // cout<<(i%k)<<endl;
        labels[i] = i%k;
    }
    for(int i=0;i<k;i++)
    {
        means[i].x = points[ind[i]].x;
        means[i].y = points[ind[i]].y;
    }

    Point* imeans = (Point*) malloc(k*sizeof(Point));
    int* ilabels = (int*) malloc(num_points*sizeof(int));
    for(int i=0;i<num_points;i++) ilabels[i] = labels[i];
    for(int i=0;i<k;i++) { imeans[i].x = means[i].x; imeans[i].y = means[i].y;} 
    printf("imeans and ilabels Allocated\n");

    Point* gpumeans;
    hipMalloc(&gpumeans, k*sizeof(Point));
    hipMemcpy(gpumeans,means,k*sizeof(Point),hipMemcpyHostToDevice); 
    printf("GPU means Allocated\n");
    
    int* gpulabels;
    hipMalloc(&gpulabels, num_points*sizeof(int));
    hipMemcpy(gpulabels,labels,num_points*sizeof(int),hipMemcpyHostToDevice); 
    printf("GPU labels Allocated\n");

    int* gpuilabels;
    hipMalloc(&gpuilabels, num_points*sizeof(int));
    hipMemcpy(gpuilabels,ilabels,num_points*sizeof(int),hipMemcpyHostToDevice); 
    printf("GPU ilabels Allocated\n");

    int iter = 1000; // maximum iteration
    float prob = 0.0001; // converging criteria
    // cpu
    float *dist = (float*)malloc(num_points*k*sizeof(float));
    printf("Dist Allocated\n");
    float* gpudist;
    hipMalloc(&gpudist, num_points*k*sizeof(float));
    printf("GPU Dist Allocated\n");

    ///////////////////////// *************** STANDARD CPU ALGORITHM *************:228******* 
    int it = 0;
    float time = 0;
    float avgout[3];
    for(int j=0;j<3;j++) avgout[j]=0;

    while(it < iter){
        float out[3];
        float ct = kmeans_cpu( points,means,labels,dist,iter,num_points,k,out);
        
        time+=ct;
        it+=1;
        
        for(int j=0;j<3;j++) avgout[j]+=out[j];
        float ch = (float) out[1];
        if( ch/num_points < prob ) break;
    }

    printf("CPU Time taken: %.6f ms\n", time/(float)it);

    for(int j=0;j<3;j++) avgout[j]/=it;
    //// Writing to file ////
    fprintf(outfile,"%f %f %f\n",avgout[0],avgout[1],avgout[2]);
    for(int i=0;i<k;i++)
    {
        fprintf(outfile,"%f %f\n",means[i].x,means[i].y);
    }
    for(int i=0;i<num_points;i++){ fprintf(outfile,"%d ",labels[i]);}
    fprintf(outfile,"\n");

    for(int i=0;i<num_points;i++){ printf("%d ",labels[i]);}
    printf("\n");
    
    ///////////////////////// *************** STANDARD GPU ALGORITHM ******************** 
    it = 0;
    time = 0;
    for(int j=0;j<3;j++) avgout[j]=0;
    while(it < iter){
            float out[3];
            float gt = kmeans_gpu(gpupoints,points,gpumeans,gpulabels,gpudist,dist,iter,num_points,k,out);

            time+=gt;
            it++;
            
            for(int j=0;j<3;j++) avgout[j]+=out[j];
            float ch = (float) out[1];
             if( ch/num_points < prob) break;
    }

     printf("GPU Time taken: %.6f ms\n", time/it);
     printf("iterations %d\n",it);
    hipMemcpy(labels,gpulabels,num_points*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(means,gpumeans,k*sizeof(Point),hipMemcpyDeviceToHost);

    for(int j=0;j<3;j++) avgout[j]/=it;
     //// Writing to file ////
    for(int i=0;i<num_points;i++){ printf("%d ",labels[i]);}
    printf("\n");
    fprintf(outfile,"%f %f %f\n",avgout[0],avgout[1],avgout[2]);
    for(int i=0;i<k;i++)
    {
        fprintf(outfile,"%f %f\n",means[i].x,means[i].y);
    }
    for(int i=0;i<num_points;i++){ fprintf(outfile,"%d ",labels[i]);}
    fprintf(outfile,"\n");
    
    ///////////////////////// *************** Reinforced CPU ALGORITHM ******************** 
    it = 0;
    time = 0;
    float* icd = (float*) malloc(k*k*sizeof(float));
    int* rid = (int*) malloc(k*k*sizeof(int));
    printf("RID and ICD Allocated\n");
    for(int j=0;j<3;j++) avgout[j]=0;

    while(it < iter){
        float out[3];
        float ct = kmeans_cpu_ineq( points,imeans,ilabels,icd,rid,iter,num_points,k,out);

        time+=ct;
        it+=1;
        for(int j=0;j<3;j++) avgout[j]+=out[j];
        float ch = (float) out[1];
        if( ch/num_points < prob) break;
    }

    printf("CPU Time taken for ineq cpu: %.6f ms\n", time/it);
    for(int i=0;i<num_points;i++){ printf("%d ",ilabels[i]);}
    printf("\n");

    for(int j=0;j<3;j++) avgout[j]/=it;
     //// Writing to file ////
    fprintf(outfile,"%f %f %f\n",avgout[0],avgout[1],avgout[2]);
    for(int i=0;i<k;i++)
    {
        fprintf(outfile,"%f %f\n",imeans[i].x,imeans[i].y);
    }
    for(int i=0;i<num_points;i++){ fprintf(outfile,"%d ",ilabels[i]);}
    fprintf(outfile,"\n");


    // Replacing imeans and ilabels as random for the GPU Algo
    for(int i=0;i<num_points;i++)
    {
        // cout<<(i%k)<<endl;
        ilabels[i] = i%k;
    }
   
    for(int i=0;i<k;i++)
    {
        imeans[i].x = points[ind[i]].x;
        imeans[i].y = points[ind[i]].y;
    }
   
    ///////////////////////// *************** Reinforced GPU ALGORITHM (Inefficient) ******************** 
    int* mcpu;
    mcpu = (int*) malloc(num_points*sizeof(int));
   
    it = 0;
    time = 0;
    for(int j=0;j<3;j++) avgout[j]=0;
    int* m;
    hipMalloc(&m,num_points*(sizeof(int)));
   
    while(it<iter){
        float out[3];
        hipMemset(m,0,num_points*sizeof(int));
        float gt = kmeans_gpu_ineq( gpupoints, imeans, gpuilabels, icd, rid, iter, num_points, k,out,m);
        hipMemcpy(mcpu,m,num_points*sizeof(int),hipMemcpyDeviceToHost);
        double c = 0;
        for(int i=0;i<num_points;i++) c+=mcpu[i];
        // printf("%f \n", c/num_points);
        time+=gt;
        it++;
        for(int j=0;j<3;j++) avgout[j]+=out[j];
        float ch = (float) out[1];
        //  printf("%f\n",ch);
        if( ch/num_points < prob) break;
    }

    printf("GPU Time taken for ineq gpu inefficient version: %.6f ms\n", time/it);
     printf("iterations %d\n",it);
    hipMemcpy(ilabels,gpuilabels,num_points*sizeof(int),hipMemcpyDeviceToHost); 
    for(int i=0;i<num_points;i++){ printf("%d ",ilabels[i]);}
    printf("\n");
    for(int j=0;j<3;j++) avgout[j]/=it;
    
     //// Writing to file ////
    fprintf(outfile,"%f %f %f\n",avgout[0],avgout[1],avgout[2]);
    for(int i=0;i<k;i++)
    {
        fprintf(outfile,"%f %f\n",imeans[i].x,imeans[i].y);
    }
    for(int i=0;i<num_points;i++){ fprintf(outfile,"%d ",ilabels[i]);}
    fprintf(outfile,"\n");

    // // Replacing imeans and ilabels as random
    // for(int i=0;i<k;i++) { num[i] = 0;}
    for(int i=0;i<num_points;i++)
    {
        // cout<<(i%k)<<endl;
        ilabels[i] = i%k;
    }
    for(int i=0;i<k;i++)
    {
        imeans[i].x = points[ind[i]].x;
        imeans[i].y = points[ind[i]].y;
    }
    // restoring gpu variables
    hipMemcpy(gpuilabels,ilabels,num_points*sizeof(int),hipMemcpyHostToDevice);
    ///////////////////////// *************** Efficient Reinforced GPU ALGORITHM ******************** 
    // Epoch 1
    printf("Efficient implementation\n");
    printf("Epoch 1\n");
    it = 0;
    time = 0;
    for(int j=0;j<3;j++) avgout[j]=0;
    
    while( it < 3)
    {
        float out[3];
        
        hipMemset(m,0,num_points*sizeof(int));

        float gt = kmeans_gpu_ineq( gpupoints, imeans, gpuilabels, icd, rid, iter, num_points, k,out,m);
        hipMemcpy(mcpu,m,num_points*sizeof(int),hipMemcpyDeviceToHost);

        // printf("mcpus\n");
        // for(int i=0;i<num_points;i++) printf("%d ",mcpu[i]);
        // printf("\n");

        float c = 0;
        
        for(int i=0;i<num_points;i++) c+=mcpu[i];
        
        time+=gt;
        it++;

        for(int j=0;j<3;j++) avgout[j]+=out[j];
        float ch = (float) out[1];

        if( ch/num_points < prob) break;
    }
    printf("GPU Time taken for ineq gpu Epoch 1: %.6f ms\n", time/it);

    vector<pair<int,int>> v;
    for(int i=0;i<num_points;i++) v.push_back(make_pair(-mcpu[i],i));
    sort(v.begin(),v.end());
    
    printf("Rearranging Points\n");
    int* inds = (int*)malloc(num_points*sizeof(int));
    int* gpuinds;
    hipMalloc(&gpuinds,num_points*sizeof(int));
    for(int i=0;i<num_points;i++)
    {
        inds[i] = v[i].second;
    }
    hipMemcpy(gpuinds,inds,num_points*sizeof(int),hipMemcpyHostToDevice);

    Point* points2 = (Point*) malloc(num_points*sizeof(Point));
    for(int i=0;i<num_points;i++) { points2[i].x = points[inds[i]].x; points2[i].y = points[inds[i]].y; }
    hipMemcpy(gpupoints,points2,num_points*sizeof(Point),hipMemcpyHostToDevice); 

    printf("GPU Points Copied\n");
    printf("Epoch 2\n");
    // it = 0;
    // time = 0;
    // for(int j=0;j<3;j++) avgout[j]=0;
    printf("Indices\n");
    for(int i=0;i<num_points;i++) printf("%d ",inds[i]);
    printf("\n");
    
    time = 0;
    it = 0;
    for(int j=0;j<3;j++) avgout[j]=0;

    while(it<iter)
    {
        float out[3];
        float gt = kmeans_gpu_ineq_eff( gpupoints,gpuinds, imeans, gpuilabels, icd, rid, iter, num_points, k,out);
        time+=gt;
        it++;
        for(int j=0;j<3;j++) avgout[j]+=out[j];
        float ch = (float) out[1];
        //  printf("ch : %f\n",ch);
        if( ch/num_points < prob) break;
    }
    printf("GPU Time taken for ineq gpu effective: %.6f ms\n", time/it);
     printf("iterations %d\n",it);
    hipMemcpy(ilabels,gpuilabels,num_points*sizeof(int),hipMemcpyDeviceToHost); 

    for(int i=0;i<num_points;i++){ printf("%d ",ilabels[i]);}
    printf("\n");
    for(int j=0;j<3;j++) avgout[j]/=it;
    
     //// Writing to file ////
    fprintf(outfile,"%f %f %f\n",avgout[0],avgout[1],avgout[2]);
    for(int i=0;i<k;i++)
    {
        fprintf(outfile,"%f %f\n",imeans[i].x,imeans[i].y);
    }
    for(int i=0;i<num_points;i++){ fprintf(outfile,"%d ",ilabels[i]);}
    fprintf(outfile,"\n");
    return 0;
}


