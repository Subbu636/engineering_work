#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <sys/time.h> 
#include "util.h"
using namespace std;
Point* read(char *filename, int* num){
    FILE *file;
    file = fopen(filename, "r");
    if (!file)  {
        printf("#cannot open input file\n");
        return NULL;
    }
    vector<Point> points;
    float x,y;
    int n = 0;
    while(fscanf(file,"%f %f",&x,&y) != EOF){
        Point p;
        p.x = x;
        p.y = y;
        points.push_back(p);
        n++;
    }
     Point* ps = (Point*) malloc(n*sizeof(Point));
     for(int i=0;i<n;i++)
     {
         ps[i] = points[i];
     }
     num[0] = n;
     return ps;
}
int main(int argc,char **argv){
    int seed = 1234;
    struct timeval t1, t2;
    char *fname = argv[1];
    if(fname == NULL)
    {
        printf("give the input\n");
    }
    int k = 20;
    Point* points;
    int p[1];
    p[0] = 0;
    points = read(fname,p);
    
    // for(int i=0;i<100;i++)
    // {
    //     printf("%f %f\n", points[i].x,points[i].y);
    // }
    // printf("%d\n", sizeof(points));
    int num_points = p[0];
    printf("Points Allocated\n");
    // return 0;
    Point* gpupoints;
    hipMalloc(&gpupoints, num_points*sizeof(Point));
    printf("GPU Points Allocated\n");
    // Point* cpoints;
    // cpoints = (Point*) malloc(n*sizeof(Point));
    hipMemcpy(gpupoints,points,num_points*sizeof(Point),hipMemcpyHostToDevice); 
    printf("GPU Points Copied\n");
    Point* means = (Point*) malloc(k*sizeof(Point));
    int* labels = (int*) malloc(num_points*sizeof(int));
    printf("Means and Labels Allocated\n");
    int num[k];
    for(int i=0;i<k;i++) { num[i] = 0;}
    
    for(int i=0;i<num_points;i++)
    {
        // cout<<(i%k)<<endl;
        labels[i] = i%k;
    }
    // printf("\n");
    
    for(int i=0;i<num_points;i++)
    {
        num[labels[i]] ++;
        means[labels[i]].x += points[i].x;
        means[labels[i]].y += points[i].y;
    }
    // for(int i=0;i<k;i++){ printf("%d ",num[i]);}
    // printf("\n");
    for(int i=0;i<k;i++)
    {
        means[i].x /= (float) num[i];
         means[i].y /= (float) num[i];
    }

    Point* imeans = (Point*) malloc(k*sizeof(Point));
    int* ilabels = (int*) malloc(num_points*sizeof(int));
    for(int i=0;i<num_points;i++) ilabels[i] = labels[i];
    for(int i=0;i<k;i++) { imeans[i].x = means[i].x; imeans[i].y = means[i].y;} 
    printf("imeans and ilabels Allocated\n");
    Point* gpumeans;
    hipMalloc(&gpumeans, k*sizeof(Point));
    hipMemcpy(gpumeans,means,k*sizeof(Point),hipMemcpyHostToDevice); 
    printf("GPU means Allocated\n");
    // for(int i=0;i<k;i++){ printf("%d ",num[i]);}
    // printf("\n");
    int* gpulabels;
    hipMalloc(&gpulabels, num_points*sizeof(int));
    hipMemcpy(gpulabels,labels,num_points*sizeof(int),hipMemcpyHostToDevice); 
    printf("GPU labels Allocated\n");
    int* gpuilabels;
    hipMalloc(&gpuilabels, num_points*sizeof(int));
    hipMemcpy(gpuilabels,ilabels,num_points*sizeof(int),hipMemcpyHostToDevice); 
    printf("GPU ilabels Allocated\n");
    // for(int i=0;i<k;i++){ printf("%d ",num[i]);}
    // printf("\n");
    //Updating above variables
    int iter = 1000;
    // cpu
    float dist[num_points*k];
    printf("Dist Allocated\n");
    float* gpudist;
    hipMalloc(&gpudist, num_points*k*sizeof(float));
    printf("GPU Dist Allocated\n");
    // for(int i=0;i<k;i++){ printf("%d ",num[i]);}
    // printf("\n");
    int i = 0;
    float time = 0;
    while(i < iter){
       
        // printf("%d \n",i);
        float ct = kmeans_cpu( points,means,labels,dist,iter,num_points,k);
        
        time+=ct;
        i+=1;
    }

    printf("CPU Time taken: %.6f ms\n", time/(float)iter);
    for(int i=0;i<num_points;i++){ printf("%d ",labels[i]);}
    printf("\n");
    // gpu
    i = 0;
    time = 0;
    while(i < iter){
            
            float gt = kmeans_gpu(gpupoints,points,gpumeans,gpulabels,gpudist,dist,iter,num_points,k);
            time+=gt;
            i++;
    }
     printf("GPU Time taken: %.6f ms\n", time/iter);
    hipMemcpy(labels,gpulabels,num_points*sizeof(int),hipMemcpyDeviceToHost); 
    for(int i=0;i<num_points;i++){ printf("%d ",labels[i]);}
    printf("\n");
   
     i = 0;
    time = 0;
    float* icd = (float*) malloc(k*k*sizeof(float));
    int* rid = (int*) malloc(k*k*sizeof(int));
    while(i < iter){
        // printf("%d \n",i);
        float ct = kmeans_cpu_ineq( points,imeans,ilabels,icd,rid,iter,num_points,k);
        time+=ct;
        i+=1;
    }

    printf("CPU Time taken for ineq cpu: %.6f ms\n", time/iter);
    for(int i=0;i<num_points;i++){ printf("%d ",ilabels[i]);}
    printf("\n");
     
    i = 0;
    time = 0;
    
    while(i<iter){
        float gt = kmeans_gpu_ineq( gpupoints, imeans, gpuilabels, icd, rid, iter, num_points, k);
        time+=gt;
        i++;
    }

    printf("GPU Time taken for ineq gpu: %.6f ms\n", time/iter);
    hipMemcpy(ilabels,gpuilabels,num_points*sizeof(int),hipMemcpyDeviceToHost); 
    for(int i=0;i<num_points;i++){ printf("%d ",ilabels[i]);}
    printf("\n");
    
    return 0;
}


