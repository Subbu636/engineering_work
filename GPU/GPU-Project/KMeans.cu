#include "hip/hip_runtime.h"
#include "util.h"

vector <Point> kmeans_cpu(vector <Point> points, int iter){
    vector <Point> means;
    // write kmeans cpu 
    return means;
} 

__global__ void distance_update(double* dist, Points* points, Points* mean)
{
    int id = (blockIdx.x*blockDim.x) + threadIdx.x;
    int i = blockIdx.x;
    int j = threadIdx.x;
    Point a = points[i];
    Point b = mean[i];
    double xdist = (a.x - b.x)*(a.x - b.x);
    double ydist = (a.y - b.y)*(a.y - b.y);
    dist[id] = xdist + ydist;
}
__global__ void label_update(int* labels,double* dist,int k,int n)
{
    int id = threadIdx.x;
    if(id < n){
        int l = 0;
        double mindist = dist[id*n]; 
        for(int i=1;i<k;i++)
        {
            if(mindist > dist[id*n + i]){
                l = i;
                mindist = dist[id*n + i];
            }
        }
        labels[id] = l;
    }
}
__global__ void centers_update(Points* means,Points* points,int* labels,int n,int k)
{
    int id = threadIdx.x;
    if(id<k){
        means[id].x = 0;
        means[id].y = 0;
        double num_points = 0;
        for(int i=0;i<n;i++){
            if(labels[i]==id)
            {
                num_points+=1;
                means[id].x += points[i].x;
                means[id].y += points[i].y;
            }
        }
        means[id].x /= num_points;
        means[id].y /= num_points;
    }
}
void kmeans_gpu(Points* points, Points* means,int* labels,double* dist, int iter, int n, int k){
    // distances size n*k 
    // write kmeans gpu  
    distance_update<<<n,k>>>(dist,points,mean);
    label_update<<<1,n>>>(labels,dist,k,n);
    centers_update<<<1,k>>>(means,points,labels,n,k);
} 