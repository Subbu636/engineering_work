#include "hip/hip_runtime.h"
#include "util.h"
#include <sys/time.h> 

void dist_update(float* dist, Point* points, Point* means, int n, int k)
{
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<k;j++)
        {
            // printf("%d %d\n",i,j);
            Point a = points[i];
            Point b = means[j];
            float xdist = (a.x - b.x)*(a.x - b.x);
            float ydist = (a.y - b.y)*(a.y - b.y);
            dist[i*k + j] = sqrt(xdist + ydist);
        }
    }
}
void label_update_cpu(int* labels, float* dist, int k,int n)
{
    for(int i=0;i<n;i++)
    {
        int l = 0;
        float mindist = dist[i*k]; 
        for(int j=1;j<k;j++)
        {
            if(mindist > dist[i*k + j]){
                l = j;
                mindist = dist[i*k + j];
            }
        }
        labels[i] = l;
    }
}
void centers_update_cpu(Point* means,Point* points,int* labels,int n,int k)
{
    for(int j=0;j<k;j++)
    {
        means[j].x = 0;
        means[j].y = 0;
        float num_points = 0;
        for(int i=0;i<n;i++){
            if(labels[i]==j)
            {
                num_points+=1;
                means[j].x += points[i].x;
                means[j].y += points[i].y;
            }
        }
        means[j].x /= num_points;
        means[j].y /= num_points;
    }
}
float kmeans_cpu(Point* points, Point* means,int* labels,float* dist, int iter, int n, int k){
    
    // write kmeans cpu 
    // printf("hey\n");
        struct timeval t1, t2;

     gettimeofday(&t1, 0);
    dist_update(dist,points,means,n,k);
    // printf("dist is fine\n");
    
    label_update_cpu(labels,dist,k,n);
         gettimeofday(&t2, 0);
    // printf("labels is fine\n");
    centers_update_cpu(means,points,labels,n,k);

     float ct = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    // for(int i=0;i<100;i++){ printf("%d ",labels[i]);}
    // printf("\n");
    return ct;
} 

__global__ void distance_update(float* dist, Point* points, Point* means, int n, int k)
{
    // int id = (blockIdx.x*blockDim.x) + threadIdx.x;
    // int i = blockIdx.x;
    for(int i=0;i<n;i++){
        int j = threadIdx.x;
        Point a = points[i];
        Point b = means[j];
        float xdist = (a.x - b.x)*(a.x - b.x);
        float ydist = (a.y - b.y)*(a.y - b.y);
        dist[i*k + j] = sqrt(xdist + ydist);
    }
}
__global__ void label_update(int* labels,float* dist,int k,int n)
{
    int id = (blockIdx.x*blockDim.x) + threadIdx.x;
    if(id < n){
        int l = 0;
        float mindist = dist[id*k]; 
        for(int i=1;i<k;i++)
        {
            if(mindist > dist[id*k + i]){
                l = i;
                mindist = dist[id*k + i];
            }
        }
        labels[id] = l;
    }
}
__global__ void centers_update(Point* means,Point* points,int* labels,int n,int k)
{
    int id = threadIdx.x;
    if(id<k){
        means[id].x = 0;
        means[id].y = 0;
        float num_points = 0;
        for(int i=0;i<n;i++){
            if(labels[i]==id)
            {
                num_points+=1;
                means[id].x += points[i].x;
                means[id].y += points[i].y;
            }
        }
        means[id].x /= num_points;
        means[id].y /= num_points;
    }
}
float kmeans_gpu(Point* points, Point* cpupoints, Point* means,int* labels,float* dist,float* cpudist, int iter, int n, int k){
    // distances size n*k 
    // write kmeans gpu  
        struct timeval t1, t2;

    
     // cpu dist
    Point cpumeans[k];
    hipMemcpy(cpumeans,means,k*sizeof(Point),hipMemcpyDeviceToHost);

     gettimeofday(&t1, 0);
    dist_update(cpudist,cpupoints,cpumeans,n,k);
     gettimeofday(&t2, 0);

    hipMemcpy(dist,cpudist,n*k*sizeof(float),hipMemcpyHostToDevice);
    float a = n/1024 + 1;
    float ct = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

     gettimeofday(&t1, 0);
    label_update<<<a,1024>>>(labels,dist,k,n);
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);
    centers_update<<<1,k>>>(means,points,labels,n,k);
    hipDeviceSynchronize();
    // gettimeofday(&t2, 0);
    ct += (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    return ct;
} 
void icd_update(Point* means, float* icd, int k)
{
    for(int i=0;i<k;i++)
    {
        for(int j=0;j<k;j++)
        {
            Point a = means[i];
            Point b = means[j];
            icd[i*k + j] = (a.x-b.x)*(a.x-b.x) + (a.y-b.y)*(a.y-b.y);
        }
    }
}
void rid_update(float* icd,int* rid,int k){
    for(int i=0;i<k;i++)
    {
        vector< pair<float,int> > v;
        for(int j=0;j<k;j++)
        {
            v.push_back(make_pair(icd[i*k+j],j));
        }
        sort(v.begin(),v.end());
        for(int j=0;j<k;j++)
        {
            rid[i*k + j] = v[j].second;
        }
    }

}
void label_update_ineq(Point* points,int* labels,Point* means,float* icd, int* rid , int n, int k){
     for(int i=0; i<n ;i++)
     {
         int curr_cent = labels[i];
         Point p = points[i];
         Point c = means[curr_cent];
         float d = (p.x-c.x)*(p.x - c.x) +  (p.y-c.y)*(p.y - c.y);
         float new_d = d;
         float new_cent = curr_cent;
         for(int j=1;j<k;j++)
         {
            int cent = rid[curr_cent*k + j];
            if (icd[curr_cent*k + cent] > 2*d )
            {
                break;
            }
            Point curr_c = means[cent];
            float curr_d = (p.x-curr_c.x)*(p.x-curr_c.x) + (p.y-curr_c.y)*(p.y-curr_c.y);
            if(curr_d < new_d)
            {
                new_d = curr_d;
                new_cent = cent;
            }
         }
         labels[i] = new_cent;
     }
}
float kmeans_cpu_ineq(Point* points,Point* means, int* labels,float* icd,int* rid,int iter,int n,int k)
{
    struct timeval t1, t2;

    gettimeofday(&t1, 0);
    icd_update(means,icd,k);
    rid_update(icd,rid,k);
    label_update_ineq(points,labels,means,icd,rid,n,k);
    gettimeofday(&t2, 0);
    centers_update_cpu(means,points,labels,n,k);
    float ct = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    return ct;
}
__global__ void label_update_ineq_gpu(Point* points,int* labels,Point* means,float* icd, int* rid,int n,int k)
{
     int t =  ceil( (float)n/ (gridDim.x * blockDim.x) );
     int s =  blockIdx.x * (n/gridDim.x) + threadIdx.x * t;
    //  printf("%d %d\n",s,s+t);
     for(int i = s;i< s+t ;i++)
     {
         if( i < n) {
            int curr_cent = labels[i];
            Point p = points[i];
            Point c = means[curr_cent];
            float d = (p.x-c.x)*(p.x - c.x) +  (p.y-c.y)*(p.y - c.y);
            float new_d = d;
            float new_cent = curr_cent;
            for(int j=1;j<k;j++)
            {
                int cent = rid[curr_cent*k + j];
                if (icd[curr_cent*k + cent] > 2*d )
                {
                    break;
                }
                Point curr_c = means[cent];
                float curr_d = (p.x-curr_c.x)*(p.x-curr_c.x) + (p.y-curr_c.y)*(p.y-curr_c.y);
                if(curr_d < new_d)
                {
                    new_d = curr_d;
                    new_cent = cent;
                }
            }
            labels[i] = new_cent;
         }
     }
}
float kmeans_gpu_ineq(Point* points, Point* means, int* labels, float* icd, int* rid, int iter, int n, int k){
    // means , rid, icd CPU
    int minGridSize, gridSize, blockSize;
    //  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,label_update_ineq, 0, 1024); 
    // Round up according to array size 
    blockSize = 1024;
    gridSize = (n + blockSize - 1) / blockSize; 

    struct timeval t1, t2;
    float* gpuicd;
    int* gpurid;
    Point* gpumeans;
    hipMalloc(&gpuicd,k*k*sizeof(float));
    hipMalloc(&gpurid,k*k*sizeof(int));
    hipMalloc(&gpumeans,k*sizeof(Point));

    gettimeofday(&t1, 0);
    icd_update(means,icd,k);
    rid_update(icd,rid,k);
        gettimeofday(&t2, 0);
float ct = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

    hipMemcpy(gpumeans,means,k*sizeof(Point),hipMemcpyHostToDevice);
    hipMemcpy(gpuicd,icd,k*k*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(gpurid,rid,k*k*sizeof(int),hipMemcpyHostToDevice);
    gettimeofday(&t1, 0);
    label_update_ineq_gpu<<<gridSize,blockSize>>>(points,labels,gpumeans,gpuicd, gpurid,n,k);
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);
    ct += (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    centers_update<<<1,k>>>(gpumeans,points,labels,n,k);   
    hipDeviceSynchronize();
    hipMemcpy(means,gpumeans,k*sizeof(Point),hipMemcpyDeviceToHost);
    
    return ct;
}