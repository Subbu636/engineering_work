#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void init(int *arr,int n){
    if(threadIdx.x < n){
        arr[threadIdx.x] = 0;
    }
}

__global__ void add(int *arr,int n){
    if(threadIdx.x < n){
        arr[threadIdx.x] += threadIdx.x;
    }
}

__global__ void print(){
    printf("GPU\n");
}

int main(){
    // cudaError_t cudaStat;
    // const int n = 10;
    // int arr[n],i;
    // int *garr;
    // cudaStat = cudaMalloc(&garr,n*sizeof(int));
    // if (cudaStat != cudaSuccess) {
    //     printf ("device memory allocation failed\n");
    // }
    // init<<<1,n>>>(garr,n);
    // cudaDeviceSynchronize();
    // add<<<1,n>>>(garr,n);
    // cudaDeviceSynchronize();
    // cudaMemcpy(arr,garr,n*sizeof(int),cudaMemcpyDeviceToHost);
    // for(i = 0;i < n;++i){
    //     printf("%d ",arr[i]);
    // }
    // printf("\n");
    print<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}