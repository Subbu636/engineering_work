#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *matrix) {
    unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
    matrix[id] = id;
}
#define x_lim = 100
#define y_lim = 100
int main() {
    const int n = 1024;
    int x[n],y[n];
    for(int i = 0;i < n;++i){
        scanf("%d%d",&x[i],&y[i]);
    }
    
    return 0;
}




