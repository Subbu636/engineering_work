// RUN: nvcc -arch=sm_50 -Wno-deprecated-gpu-targets hello.cu

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
 printf("%d\n",threadIdx.x);
}
int main() {
 dkernel<<<1, 100>>>();
 hipDeviceSynchronize();
 return 0;
}

// xorg-edgers/ppa