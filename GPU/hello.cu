// RUN: nvcc -arch=sm_50 -Wno-deprecated-gpu-targets hello.cu

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <bits/stdc++.h>

__global__ void dkernel(hipDoubleComplex* d_cplx) {
    int id = threadIdx.x;
    if (id <= 3){
        printf("%f\n",d_cplx[id]);
    }
}

int main() {
    std::vector<int> h_cplx;
    // h_cplx.push_back(1.0);
    // h_cplx.push_back(2.3);
    // h_cplx.push_back(3.4);
    // cuDoubleComplex* d_cplx;
    // cudaMalloc(&d_cplx, h_cplx.size()*sizeof(cuDoubleComplex)); 
    // cudaMemcpy(d_cplx, h_cplx.data(), h_cplx.size()*sizeof(cuDoubleComplex), cudaMemcpyHostToDevice);
    // dkernel<<<2, 30>>>(d_cplx);
    // cudaDeviceSynchronize();
    return 0;
}

// xorg-edgers/ppa