#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void init(int *arr,int n){
    if(threadIdx.x < n){
        arr[threadIdx.x] = 0;
    }
}

__global__ void add(int *arr,int n){
    if(threadIdx.x < n){
        arr[threadIdx.x] += threadIdx.x;
    }
}

int main(){
    const int n = 10;
    int arr[n],i;
    int *garr;
    hipMalloc(&garr,n*sizeof(int));
    init<<<1,n>>>(garr,n);
    hipDeviceSynchronize();
    add<<<1,n>>>(garr,n);
    hipDeviceSynchronize();
    hipMemcpy(arr,garr,n*sizeof(int),hipMemcpyDeviceToHost);
    for(i = 0;i < n;++i){
        printf("%d ",arr[i]);
    }
    printf("\n");
    return 0;
}